#include "hip/hip_runtime.h"
/*********************************************************************************
 * contractions.cu
 *
 * Sat Jul  2 11:19:43 CEST 2011
 *
 * PURPOSE:
 * TODO:
 * DONE:
 * CHANGES:
 *********************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#ifdef MPI
#  include <mpi.h>
#  include <fftw_mpi.h>
#else
#  include <fftw.h>
#endif
#include <getopt.h>

#define MAIN_PROGRAM
extern "C"
{
#include "cvc_complex.h"
#include "cvc_linalg.h"
#include "global.h"
#include "cvc_geometry.h"
#include "cvc_utils.h"
#include "contractions_io.h"
#include "read_input_parser.h"
#include "contractions.h"
}

__constant__ int devT, devL;
__constant__ float devMu, devMq;
__constant__ float dev_cvc_coeff[2304];

/*************************************************************
 * the kernel for contract cvc
 *************************************************************/
__global__ void cvc_kernel (float*cvc_out, float*ct_out, unsigned int N) {

  int j0, j1, j2, j3, i0, i1, i2, i3;
  unsigned int L1, L2, L3, V4, imu, inu, icount, rest;
  // unsigned int tid = threadIdx.x;
  unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int isigma_mu, isigma_nu, ilambda, ikappa;
  float2 sp[6], sq[6];
  float p[4], phalf[4], sinp[4], cosp[4], sinphalf[4], sinq[4], sinqhalf[4];
  float q[4], qhalf[4], k[4];
  // float khalf[4];
  float2 phase[2][2];
  float ftmp;
  float2 cvc_tmp[16], f2tmp, f2tmp2, counter_term[4];
  float fTinv, fLinv;
  float aMp, aK2p, denomp;
  float aMq, aK2q, denomq;

  // get external momentum k from idx
  L1  = devL;
  L2 = L1 * L1;
  L3 = L2 * L1;
  V4 = L3 * devT;

  if(idx < N) {
    // initialize
    counter_term[0].x = 0.; counter_term[0].y = 0.;    
    counter_term[1].x = 0.; counter_term[1].y = 0.;    
    counter_term[2].x = 0.; counter_term[2].y = 0.;    
    counter_term[3].x = 0.; counter_term[3].y = 0.;    
    
    ftmp = -3.;  
    cvc_tmp[ 0].x = ftmp; cvc_tmp[ 0].y = ftmp;
    cvc_tmp[ 1].x = ftmp; cvc_tmp[ 1].y = ftmp;
    cvc_tmp[ 2].x = ftmp; cvc_tmp[ 2].y = ftmp;
    cvc_tmp[ 3].x = ftmp; cvc_tmp[ 3].y = ftmp;
    cvc_tmp[ 4].x = ftmp; cvc_tmp[ 4].y = ftmp;
    cvc_tmp[ 5].x = ftmp; cvc_tmp[ 5].y = ftmp;
    cvc_tmp[ 6].x = ftmp; cvc_tmp[ 6].y = ftmp;
    cvc_tmp[ 7].x = ftmp; cvc_tmp[ 7].y = ftmp;
    cvc_tmp[ 8].x = ftmp; cvc_tmp[ 8].y = ftmp;
    cvc_tmp[ 9].x = ftmp; cvc_tmp[ 9].y = ftmp;
    cvc_tmp[10].x = ftmp; cvc_tmp[10].y = ftmp;
    cvc_tmp[11].x = ftmp; cvc_tmp[11].y = ftmp;
    cvc_tmp[12].x = ftmp; cvc_tmp[12].y = ftmp;
    cvc_tmp[13].x = ftmp; cvc_tmp[13].y = ftmp;
    cvc_tmp[14].x = ftmp; cvc_tmp[14].y = ftmp;
    cvc_tmp[15].x = ftmp; cvc_tmp[15].y = ftmp;

    ftmp = -(float)idx;
    cvc_out[_GWI( 0,idx,V4)  ] = ftmp; cvc_out[_GWI( 0,idx,V4)+1] = ftmp;
    cvc_out[_GWI( 1,idx,V4)  ] = ftmp; cvc_out[_GWI( 1,idx,V4)+1] = ftmp;
    cvc_out[_GWI( 2,idx,V4)  ] = ftmp; cvc_out[_GWI( 2,idx,V4)+1] = ftmp;
    cvc_out[_GWI( 3,idx,V4)  ] = ftmp; cvc_out[_GWI( 3,idx,V4)+1] = ftmp;
    cvc_out[_GWI( 4,idx,V4)  ] = ftmp; cvc_out[_GWI( 4,idx,V4)+1] = ftmp;
    cvc_out[_GWI( 5,idx,V4)  ] = ftmp; cvc_out[_GWI( 5,idx,V4)+1] = ftmp;
    cvc_out[_GWI( 6,idx,V4)  ] = ftmp; cvc_out[_GWI( 6,idx,V4)+1] = ftmp;
    cvc_out[_GWI( 7,idx,V4)  ] = ftmp; cvc_out[_GWI( 7,idx,V4)+1] = ftmp;
    cvc_out[_GWI( 8,idx,V4)  ] = ftmp; cvc_out[_GWI( 8,idx,V4)+1] = ftmp;
    cvc_out[_GWI( 9,idx,V4)  ] = ftmp; cvc_out[_GWI( 9,idx,V4)+1] = ftmp;
    cvc_out[_GWI(10,idx,V4)  ] = ftmp; cvc_out[_GWI(10,idx,V4)+1] = ftmp;
    cvc_out[_GWI(11,idx,V4)  ] = ftmp; cvc_out[_GWI(11,idx,V4)+1] = ftmp;
    cvc_out[_GWI(12,idx,V4)  ] = ftmp; cvc_out[_GWI(12,idx,V4)+1] = ftmp;
    cvc_out[_GWI(13,idx,V4)  ] = ftmp; cvc_out[_GWI(13,idx,V4)+1] = ftmp;
    cvc_out[_GWI(14,idx,V4)  ] = ftmp; cvc_out[_GWI(14,idx,V4)+1] = ftmp;
    cvc_out[_GWI(15,idx,V4)  ] = ftmp; cvc_out[_GWI(15,idx,V4)+1] = ftmp;

    j0 = idx / L3;
    icount = idx - L3*j0;
    j1 = icount / L2;
    icount = icount - L2*j1;
    j2 = icount / L1;
    j3 = icount  - j2*L1;

    fTinv = 2. * _PI / (float)( devT );
    fLinv = 2. * _PI / (float)( devL );

    k[0] = (float)(j0) * fTinv;
    k[1] = (float)(j1) * fLinv;
    k[2] = (float)(j2) * fLinv;
    k[3] = (float)(j3) * fLinv;
/*
    khalf[0] = 0.5 * k[0];
    khalf[1] = 0.5 * k[1];
    khalf[2] = 0.5 * k[2];
    khalf[3] = 0.5 * k[3];
*/


    if(idx==102) {
      counter_term[0].x = fTinv; counter_term[0].y = fLinv;
      counter_term[1].x = k[0];  counter_term[1].y = k[1];
      counter_term[2].x = k[2];  counter_term[2].y = k[3];
      counter_term[3].x = (float)N; counter_term[3].y = (float)V4;
    } 


    // loop on internal momentum p (summation)
    i0=0; i1=0; i2=0; i3=0;
    for(icount=0; icount<V4; icount++) {

      p[0] = ((float)(i0)  + 0.5) * fTinv;
      phalf[0] = p[0] * 0.5;
      q[0] = ( ( (float)(i0) + (float)(j0) ) + 0.5 ) * fTinv;
      qhalf[0] = q[0] * 0.5;

      sinp[0]     = sin( p[0] );
      cosp[0]     = cos( p[0] );
      sinphalf[0] = sin( phalf[0] );
      sinq[0]     = sin( q[0] );
      sinqhalf[0] = sin( qhalf[0] );

      p[1] = (float)(i1)          * fLinv;
      phalf[1] = p[1] * 0.5;
      q[1] = ( (float)(i1) + (float)(j1) ) * fLinv;
      qhalf[1] = q[1] * 0.5;

      sinp[1]     = sin( p[1] );
      cosp[1]     = cos( p[1] );
      sinphalf[1] = sin( phalf[1] );
      sinq[1]     = sin( q[1] );
      sinqhalf[1] = sin( qhalf[1] );
  
      p[2] = (float)(i2)          * fLinv;
      phalf[2] = p[2] * 0.5;
      q[2] = ( (float)(i2) + (float)(j2) ) * fLinv;
      qhalf[2] = q[2] * 0.5;
  
      sinp[2]     = sin( p[2] );
      cosp[2]     = cos( p[2] );
      sinphalf[2] = sin( phalf[2] );
      sinq[2]     = sin( q[2] );
      sinqhalf[2] = sin( qhalf[2] );

      p[3] = (float)(i3) * fLinv;
      phalf[3] = p[3] * 0.5;
      q[3] = ( (float)(i3) + (float)(j3) ) * fLinv;
      qhalf[3] = q[3] * 0.5;

      sinp[3]     = sin( p[3] );
      cosp[3]     = cos( p[3] );
      sinphalf[3] = sin( phalf[3] );
      sinq[3]     = sin( q[3] );
      sinqhalf[3] = sin( qhalf[3] );
  
      aMp = devMq + 2. * (_SQR(sinphalf[0]) + _SQR(sinphalf[1]) + _SQR(sinphalf[2]) + _SQR(sinphalf[3]));

      aMq = devMq + 2. * (_SQR(sinqhalf[0]) + _SQR(sinqhalf[1]) + _SQR(sinqhalf[2]) + _SQR(sinqhalf[3]));

      aK2p = _SQR(sinp[0]) + _SQR(sinp[1]) + _SQR(sinp[2]) + _SQR(sinp[3]);

      aK2q = _SQR(sinq[0]) + _SQR(sinq[1]) + _SQR(sinq[2]) + _SQR(sinq[3]);

      denomp = 1. / ( aK2p + aMp*aMp + devMu*devMu );

      denomq = 1. / ( aK2q + aMq*aMq + devMu*devMu );

      sp[0].y = -sinp[0] * denomp;
      sp[1].y = -sinp[1] * denomp;
      sp[2].y = -sinp[2] * denomp;
      sp[3].y = -sinp[3] * denomp;
      sp[4].x =  aMp     * denomp;
      sp[5].y = -devMu   * denomp;

      sq[0].y = -sinq[0] * denomq;
      sq[1].y = -sinq[1] * denomq;
      sq[2].y = -sinq[2] * denomq;
      sq[3].y = -sinq[3] * denomq;
      sq[4].x =  aMq     * denomq;
      sq[5].y = -devMu   * denomq;

      _dev_set_phase(phase,p,k,0,0);
      cvc_tmp[0]. x += 1.;
      cvc_tmp[0].y += -2.;
/*
      _cvc_accum( cvc_tmp[ 0], 0, 0, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
*/
/*
      _dev_set_phase( phase, p, k, 0, 1 );
      _cvc_accum( cvc_tmp[ 1], 0, 1, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
      _dev_set_phase( phase, p, k, 0, 2 );
      _cvc_accum( cvc_tmp[ 2], 0, 2, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
      _dev_set_phase( phase, p, k, 0, 3 );
      _cvc_accum( cvc_tmp[ 3], 0, 3, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
      _dev_set_phase( phase, p, k, 1, 0 );
      _cvc_accum( cvc_tmp[ 4], 1, 0, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
*/
/*
      _dev_set_phase( phase, p, k, 1, 1 );
      _cvc_accum( cvc_tmp[ 5], 1, 1, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
*/
/*
      _dev_set_phase( phase, p, k, 1, 2 );
      _cvc_accum( cvc_tmp[ 6], 1, 2, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
      _dev_set_phase( phase, p, k, 1, 3 );
      _cvc_accum( cvc_tmp[ 7], 1, 3, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
      _dev_set_phase( phase, p, k, 2, 0 );
      _cvc_accum( cvc_tmp[ 8], 2, 0, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
      _dev_set_phase( phase, p, k, 2, 1 );
      _cvc_accum( cvc_tmp[ 9], 2, 1, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
*/
/*
      _dev_set_phase( phase, p, k, 2, 2 );
      _cvc_accum( cvc_tmp[10], 2, 2, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
*/
/*
      _dev_set_phase( phase, p, k, 2, 3 );
      _cvc_accum( cvc_tmp[11], 2, 3, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
      _dev_set_phase( phase, p, k, 3, 0 );
      _cvc_accum( cvc_tmp[12], 3, 0, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
      _dev_set_phase( phase, p, k, 3, 1 );
      _cvc_accum( cvc_tmp[13], 3, 1, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
      _dev_set_phase( phase, p, k, 3, 2 );
      _cvc_accum( cvc_tmp[14], 3, 2, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
*/
/*
      _dev_set_phase( phase, p, k, 3, 3 );
      _cvc_accum( cvc_tmp[15], 3, 3, dev_cvc_coeff, phase, sp, sq, f2tmp, f2tmp2 );
      //--------------------------------------------------------------------
*/
/*
      if( idx == 0 ){
        counter_term[0].x +=  sinp[0] * sp[0].y + sp[4].x * cosp[0];
        counter_term[0].y += -sinp[0] * sp[0].x + sp[4].y * cosp[0];
        counter_term[1].x +=  sinp[1] * sp[1].y + sp[4].x * cosp[1];
        counter_term[1].y += -sinp[1] * sp[1].x + sp[4].y * cosp[1];
        counter_term[2].x +=  sinp[2] * sp[2].y + sp[4].x * cosp[2];
        counter_term[2].y += -sinp[2] * sp[2].x + sp[4].y * cosp[2];
        counter_term[3].x +=  sinp[3] * sp[3].y + sp[4].x * cosp[3];
        counter_term[3].y += -sinp[3] * sp[3].x + sp[4].y * cosp[3];
      }
*/
      // increase the coordinates i0,...,i3
      i3 += 1;    rest = (i3==L1);   i3 -= rest*L1;
      i2 += rest; rest = (i2==L1);   i2 -= rest*L1;
      i1 += rest; rest = (i1==L1);   i1 -= rest*L1;
      i0 += rest; rest = (i0==devT); i0 -= rest*devT;

    }  // loop on icount

    // normalisation
    ftmp = 0.25 * _NSPIN * _NCOLOR / ( (float)(devT) * (float)(L1) * (float)(L1) * (float)(L1) );

    cvc_out[_GWI( 0,idx,V4)  ] = -cvc_tmp[ 0].x*ftmp; cvc_out[_GWI( 0,idx,V4)+1] = -cvc_tmp[ 0].y*ftmp;
    cvc_out[_GWI( 1,idx,V4)  ] = -cvc_tmp[ 1].x*ftmp; cvc_out[_GWI( 1,idx,V4)+1] = -cvc_tmp[ 1].y*ftmp;
    cvc_out[_GWI( 2,idx,V4)  ] = -cvc_tmp[ 2].x*ftmp; cvc_out[_GWI( 2,idx,V4)+1] = -cvc_tmp[ 2].y*ftmp;
    cvc_out[_GWI( 3,idx,V4)  ] = -cvc_tmp[ 3].x*ftmp; cvc_out[_GWI( 3,idx,V4)+1] = -cvc_tmp[ 3].y*ftmp;
    cvc_out[_GWI( 4,idx,V4)  ] = -cvc_tmp[ 4].x*ftmp; cvc_out[_GWI( 4,idx,V4)+1] = -cvc_tmp[ 4].y*ftmp;
    cvc_out[_GWI( 5,idx,V4)  ] = -cvc_tmp[ 5].x*ftmp; cvc_out[_GWI( 5,idx,V4)+1] = -cvc_tmp[ 5].y*ftmp;
    cvc_out[_GWI( 6,idx,V4)  ] = -cvc_tmp[ 6].x*ftmp; cvc_out[_GWI( 6,idx,V4)+1] = -cvc_tmp[ 6].y*ftmp;
    cvc_out[_GWI( 7,idx,V4)  ] = -cvc_tmp[ 7].x*ftmp; cvc_out[_GWI( 7,idx,V4)+1] = -cvc_tmp[ 7].y*ftmp;
    cvc_out[_GWI( 8,idx,V4)  ] = -cvc_tmp[ 8].x*ftmp; cvc_out[_GWI( 8,idx,V4)+1] = -cvc_tmp[ 8].y*ftmp;
    cvc_out[_GWI( 9,idx,V4)  ] = -cvc_tmp[ 9].x*ftmp; cvc_out[_GWI( 9,idx,V4)+1] = -cvc_tmp[ 9].y*ftmp;
    cvc_out[_GWI(10,idx,V4)  ] = -cvc_tmp[10].x*ftmp; cvc_out[_GWI(10,idx,V4)+1] = -cvc_tmp[10].y*ftmp;
    cvc_out[_GWI(11,idx,V4)  ] = -cvc_tmp[11].x*ftmp; cvc_out[_GWI(11,idx,V4)+1] = -cvc_tmp[11].y*ftmp;
    cvc_out[_GWI(12,idx,V4)  ] = -cvc_tmp[12].x*ftmp; cvc_out[_GWI(12,idx,V4)+1] = -cvc_tmp[12].y*ftmp;
    cvc_out[_GWI(13,idx,V4)  ] = -cvc_tmp[13].x*ftmp; cvc_out[_GWI(13,idx,V4)+1] = -cvc_tmp[13].y*ftmp;
    cvc_out[_GWI(14,idx,V4)  ] = -cvc_tmp[14].x*ftmp; cvc_out[_GWI(14,idx,V4)+1] = -cvc_tmp[14].y*ftmp;
    cvc_out[_GWI(15,idx,V4)  ] = -cvc_tmp[15].x*ftmp; cvc_out[_GWI(15,idx,V4)+1] = -cvc_tmp[15].y*ftmp;

    ftmp *= 4.;
    // if(idx==0)
    if(idx==102)
    {
      ct_out[0] = -counter_term[0].x * ftmp;
      ct_out[1] = -counter_term[0].y * ftmp;
      ct_out[2] = -counter_term[1].x * ftmp;
      ct_out[3] = -counter_term[1].y * ftmp;
      ct_out[4] = -counter_term[2].x * ftmp;
      ct_out[5] = -counter_term[2].y * ftmp;
      ct_out[6] = -counter_term[3].x * ftmp;
      ct_out[7] = -counter_term[3].y * ftmp;
    }


  }  // of if idx < N

}

/**********************************************************************
 * main program
 **********************************************************************/
int main(int argc, char **argv) {
    
  // int status;
  int c, filename_set=0, verbose=0;
  int mu, nu, x0, x1, x2, x3, ix;
  int imu, inu, isigma_mu, isigma_nu, ikappa, ilambda;
  unsigned int threadsPerBlock, blocksPerGrid;
  int i;

  double delta_mn, delta_mk, delta_nk, delta_ml, delta_nl, delta_lk;
  double sigma_mu, sigma_nu;
  float cvc_coeff[2304], phase[4];
  double *dptr = NULL;
  float *fptr  = NULL;
  // const double CVC_EPS = 5.e-15;

  void *cvc=NULL, *counter_term;
  // float WI_check;
  float ftmp;
  complex w, w1;

  char filename[80], contype[200];

  float *dev_cvc, *dev_ct;
  hipDeviceProp_t prop;

  while ((c = getopt(argc, argv, "h?f:v:")) != -1) {
    switch (c) {
      case 'f':
        strcpy(filename, optarg);
        filename_set=1;
        break;
      case 'v':
        verbose = atoi( optarg );
        fprintf(stdout, "\n# [] using verbose mode %d\n", verbose);
        break;
      default:
        //usage();
        break;
    }
  }

  /* get the time stamp */
  g_the_time = time(NULL);

  /* set the default values */
  if(filename_set==0) strcpy(filename, "cvc.input");
  fprintf(stdout, "# Reading input from file %s\n", filename);
  read_input_parser(filename);

  T = T_global;
  L = LX;

  if(init_geometry() != 0) {
    fprintf(stderr, "ERROR from init_geometry\n");
    exit(1);
  }

  geometry();

  /***********************************************
   * device properties
   ***********************************************/
  HANDLE_ERROR( hipGetDevice(&c) );
  HANDLE_ERROR(hipGetDeviceProperties(&prop, c) );
  fprintf(stdout, "\n--- General info for device no. %d\n", c);
  fprintf(stdout, "Name: %s\n", prop.name);
  fprintf(stdout, "Compute capability: %d.%d\n", prop.major, prop.minor);
  printf("Clock rate: %d\n", prop.clockRate);
  printf("Device copy overlap: ");
  if(prop.deviceOverlap) {
    printf("Enabled\n");
  } else {
    printf("Disabled\n");
  }
  printf("Kernel execution timeout: ");
  if(prop.kernelExecTimeoutEnabled) {
    printf("Enabled\n");
  } else {
    printf("Disabled\n");
  }
  printf("\n--- Memory info for device no. %d\n", c);
  printf("Total global mem: %ld\n", prop.totalGlobalMem);
  printf("Total constant mem: %ld\n", prop.totalConstMem);
  printf("Max mem pitch: %ld\n", prop.memPitch);
  printf("Texture alignment: %ld\n", prop.textureAlignment);
  printf("\n--- MP info for device no. %d\n", c);
  printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
  printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
  printf("Registers mem per mp: %d\n", prop.regsPerBlock);
  printf("Threads in warp: %d\n", prop.warpSize);
  printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
  printf("Max thread dimension: (%d, %d, %d)\n", prop.maxThreadsDim[0],
      prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
  printf("Max grid dimension: (%d, %d, %d)\n", prop.maxGridSize[0],
      prop.maxGridSize[1], prop.maxGridSize[2]);
  printf("\n\n");

  /***********************************************
   * set number of threads and blocks
   ***********************************************/
  threadsPerBlock        = THREADS_PER_BLOCK;
  blocksPerGrid          = (VOLUME + threadsPerBlock-1)/threadsPerBlock;
  fprintf(stdout, "\n# [contractions] number threads per block: %u\n", threadsPerBlock);
  fprintf(stdout, "\n# [contractions] number blocks per grid  : %u\n", blocksPerGrid);

  // allocate memory for cvc
  cvc = calloc( 32*VOLUME, sizeof(double) );
  counter_term = calloc( 8, sizeof(double) );
  if( cvc == NULL || counter_term==NULL) {
    fprintf(stderr, "\nError, could not alloc cvc\n");
    exit(2);
  }

  /***************************
   * initialize on host
   ***************************/
 
  for(imu=0;imu<2304;imu++) cvc_coeff[imu] = 0.;
 
  // set the coefficients for the correlation functions
  for(imu=0; imu<4;imu++) {
  for(inu=0; inu<4;inu++) {
    delta_mn = (float)(imu==inu);

    for(isigma_mu=0; isigma_mu<2;isigma_mu++) {
    for(isigma_nu=0; isigma_nu<2;isigma_nu++) {

      sigma_mu =  2.*isigma_mu-1.;
      sigma_nu =  2.*isigma_nu-1.;

      // C_4_4
      cvc_coeff[ _CVC_COEFF_IDX(imu, inu, isigma_mu, isigma_nu, 4, 4) ] = delta_mn + sigma_mu*sigma_nu;

      // C_4_5, C_5_4, C_l_5, C_5_k
      // all 0
      
      // C_5_5
      cvc_coeff[ _CVC_COEFF_IDX(imu, inu, isigma_mu, isigma_nu, 5, 5) ] = -delta_mn + sigma_mu*sigma_nu;

      // C_4_k
      for(ikappa=0;ikappa<4;ikappa++) {
        delta_mk = (float)( imu == ikappa) ;
        delta_nk = (float)( inu == ikappa );
        cvc_coeff[ _CVC_COEFF_IDX(imu, inu, isigma_mu, isigma_nu, 4, ikappa) ] = delta_mk*sigma_nu + delta_nk*sigma_mu;
      }

      // C_l_4
      for( ilambda=0; ilambda<4;ilambda++) {
        delta_ml = (float)(imu==ilambda); 
        delta_nl = (float)(inu==ilambda);
        cvc_coeff[ _CVC_COEFF_IDX(imu, inu, isigma_mu, isigma_nu, ilambda, 4) ] = delta_ml*sigma_nu + delta_nl*sigma_mu;
      }

        // C_l_k
      for(ilambda=0; ilambda<4;ilambda++) {
      for(ikappa=0;  ikappa<4; ikappa++ ) {
        //*************************************
        //*************************************
        delta_ml = (float)(imu==ilambda);
        // ************************************
        delta_mk = (float)(imu==ikappa);
        // ************************************
        // ************************************
        delta_nl = (float)(inu==ilambda);
        // ************************************
        delta_nk = (float)(inu==ikappa);
        // ************************************
        // ************************************
        delta_lk = (float)(ilambda==ikappa);
        // ************************************
        // ************************************

        cvc_coeff[ _CVC_COEFF_IDX(imu, inu, isigma_mu, isigma_nu, ilambda, ikappa) ] = \
            delta_ml*delta_nk - delta_mn*delta_lk + delta_mk*delta_nl + delta_lk*sigma_mu*sigma_nu;

      }}

    }}  // of isigma_mu, isigma_nu

  }} // of imu, inu

  /**************************************************************************
   * test: print the matrix cvc_coeff
   **************************************************************************/
  if(verbose > 0) {
    for(imu=0;imu<4;imu++) {
    for(inu=0;inu<4;inu++) {
      fprintf(stdout, "# ---------------------------------------------------------------\n");
      fprintf(stdout, "# imu = %d; inu = %d\n", imu, inu);
      for(isigma_mu=0;isigma_mu<2;isigma_mu++) {
      for(isigma_nu=0;isigma_nu<2;isigma_nu++) {
        fprintf(stdout, "# ---------------------------------------------------------------\n");
        fprintf(stdout, "#\t sigma_mu = %e; sigma_nu = %e\n", 2.*isigma_mu-1., 2.*isigma_nu-1.);
        for(ilambda=0;ilambda<6;ilambda++) {
          fprintf(stdout, "%5.3f\t%5.3f\t%5.3f\t%5.3f\t%5.3f\t%5.3f\n",
              cvc_coeff[ _CVC_COEFF_IDX(imu, inu, isigma_mu, isigma_nu, ilambda, 0) ],
              cvc_coeff[ _CVC_COEFF_IDX(imu, inu, isigma_mu, isigma_nu, ilambda, 1) ],
              cvc_coeff[ _CVC_COEFF_IDX(imu, inu, isigma_mu, isigma_nu, ilambda, 2) ],
              cvc_coeff[ _CVC_COEFF_IDX(imu, inu, isigma_mu, isigma_nu, ilambda, 3) ],
              cvc_coeff[ _CVC_COEFF_IDX(imu, inu, isigma_mu, isigma_nu, ilambda, 4) ],
              cvc_coeff[ _CVC_COEFF_IDX(imu, inu, isigma_mu, isigma_nu, ilambda, 5) ] );
        }
      }}
    }} 
  }
  /***************************************
   * allocate fields, initialize on device
   ***************************************/
  HANDLE_ERROR( hipMalloc(&dev_cvc, 32*VOLUME*sizeof(float)) );
  HANDLE_ERROR( hipMalloc(&dev_ct, 8*sizeof(float)) );

  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( "devT"), &T, sizeof(int), 0, hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( "devL"), &L, sizeof(int), 0, hipMemcpyHostToDevice) );
  ftmp = (float)g_mu;
  fprintf(stdout, "# [] using mu = %f\n", ftmp);
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( "devMu"), &ftmp, sizeof(float), 0, hipMemcpyHostToDevice) );
  ftmp = (float)( 1. / (2. * g_kappa) - 4. );
  fprintf(stdout, "# [] using mq = %f\n", ftmp);
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( "devMq"), &ftmp, sizeof(float), 0, hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( "dev_cvc_coeff"), cvc_coeff, 2304*sizeof(float), 0, hipMemcpyHostToDevice) );
//  HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dev_cvc_coeff), cvc_coeff, sizeof(cvc_coeff)) );

  /*************************
   * call kernel
   *************************/
  cvc_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_cvc, dev_ct, VOLUME);

  HANDLE_ERROR( hipMemcpy(cvc, dev_cvc, 32*VOLUME*sizeof(float), hipMemcpyDeviceToHost) );
  HANDLE_ERROR( hipMemcpy(counter_term, dev_ct, 8*sizeof(float), hipMemcpyDeviceToHost) );

  fprintf(stdout, "\n# [] float counter terms:\n");
  fptr = (float*)counter_term; 
  fprintf(stdout, "\t%d\t%f\t%f\n", 0, fptr[0], fptr[1]);
  fprintf(stdout, "\t%d\t%f\t%f\n", 1, fptr[2], fptr[3]);
  fprintf(stdout, "\t%d\t%f\t%f\n", 2, fptr[4], fptr[5]);
  fprintf(stdout, "\t%d\t%f\t%f\n", 3, fptr[6], fptr[7]);

  // cast to double precision
  dptr = (double*)cvc;
  fptr = (float*)cvc;
/*
  for(ix=0;ix<VOLUME;ix++) {
    for(mu=0;mu<16;mu++) {
      fprintf(stdout, "%d\t%d\t%f\t%f\n", ix, mu, fptr[_GWI(mu,ix,VOLUME)], fptr[_GWI(mu,ix,VOLUME)+1]);
    }
  }
*/
  for(i=32*VOLUME-1;i>=0;i--) dptr[i] = (double)fptr[i];


  dptr = (double*)counter_term;
  fptr = (float*)counter_term;
  for(i=7;i>=0;i--) dptr[i] = (double)fptr[i];

  /*********************************************
   * add phase factors, subtract counter term
   *********************************************/
#ifdef _UNDEF
  for(mu=0; mu<4; mu++) {
    double *phi = (double*)cvc + _GWI(5*mu,0,VOLUME);

    for(x0=0; x0<T; x0++) {
      phase[0] = 2. * (double)(x0) * M_PI / (double)T_global;
    for(x1=0; x1<LX; x1++) {
      phase[1] = 2. * (double)(x1) * M_PI / (double)LX;
    for(x2=0; x2<LY; x2++) {
      phase[2] = 2. * (double)(x2) * M_PI / (double)LY;
    for(x3=0; x3<LZ; x3++) {
      phase[3] = 2. * (double)(x3) * M_PI / (double)LZ;
      ix = g_ipt[x0][x1][x2][x3];
      phi[2*ix  ] = - ((double*)counter_term)[2*mu  ];
      phi[2*ix+1] = - ((double*)counter_term)[2*mu+1];
    }}}}
  }  /* of mu */

  for(mu=0; mu<3; mu++) {
  for(nu=mu+1; nu<4; nu++) {
    double *phi = (double*)cvc + _GWI(4*mu+nu,0,VOLUME);
    double *chi = (double*)cvc + _GWI(4*nu+mu,0,VOLUME);

    for(x0=0; x0<T; x0++) {
      phase[0] =  (double)(x0) * M_PI / (double)T_global;
    for(x1=0; x1<LX; x1++) {
      phase[1] =  (double)(x1) * M_PI / (double)LX;
    for(x2=0; x2<LY; x2++) {
      phase[2] =  (double)(x2) * M_PI / (double)LY;
    for(x3=0; x3<LZ; x3++) {
      phase[3] =  (double)(x3) * M_PI / (double)LZ;
      ix = g_ipt[x0][x1][x2][x3];
      w.re =  cos( phase[mu] - phase[nu] );
      w.im =  sin( phase[mu] - phase[nu] );
      _co_eq_co_ti_co(&w1,(complex*)( phi+2*ix ), &w);
      phi[2*ix  ] = w1.re;
      phi[2*ix+1] = w1.im;

      w.re =  cos( phase[nu] - phase[mu] );
      w.im =  sin( phase[nu] - phase[mu] );
      _co_eq_co_ti_co(&w1,(complex*)( chi+2*ix ), &w);
      chi[2*ix  ] = w1.re;
      chi[2*ix+1] = w1.im;
    }}}}
  }}  /* of mu and nu */
#endif
  // write to file
  sprintf(filename, "pi_L%.2dT%.2d_mu%6.4f", L, T, g_mu);
  sprintf(contype, "tree-level vacuum polarization");
  write_lime_contraction((double*)cvc, filename, 64, 16, contype, Nconf, 0);

  sprintf(filename, "pi_L%.2dT%.2d_mu%6.4f.ascii", L, T, g_mu);
  write_contraction((double*)cvc, NULL, filename, 16, 2, 0); 

  dptr = (double*)counter_term;
  fprintf(stdout, "\n# [] counter terms:\n");
  fprintf(stdout, "\t%d\t%e\t%e\n", 0, dptr[0], dptr[1]);
  fprintf(stdout, "\t%d\t%e\t%e\n", 0, dptr[2], dptr[3]);
  fprintf(stdout, "\t%d\t%e\t%e\n", 0, dptr[4], dptr[5]);
  fprintf(stdout, "\t%d\t%e\t%e\n", 0, dptr[6], dptr[7]);
#ifdef _UNDEF 
#endif
  /*************************************
   * free and finalize
   *************************************/
  if( cvc!=NULL ) free(cvc);
  if( counter_term!=NULL ) free(counter_term);
  hipFree( dev_cvc );
  hipFree( dev_ct );
  g_the_time = time(NULL);
  fprintf(stdout, "\n# [contractions] %s# [contractions] end of run\n", ctime(&g_the_time));
  fprintf(stderr, "\n# [contractions] %s# [contractions] end of run\n", ctime(&g_the_time));

  return(0);
}
