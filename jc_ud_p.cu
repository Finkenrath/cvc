#include "hip/hip_runtime.h"
/*********************************************************************************
 * jc_ud_p_gpu.cu
 *
 * Wed Sep 22 10:21:53 CEST 2010
 *
 * PURPOSE:
 * TODO:
 * DONE:
 * CHANGES:
 *********************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#ifdef MPI
#  include <mpi.h>
#  include <fftw_mpi.h>
#else
#  include <fftw.h>
#endif
#include <getopt.h>

#define MAIN_PROGRAM
extern "C" 
{
#include "cvc_complex.h"
#include "cvc_linalg.h"
#include "global.h"
#include "cvc_geometry.h"
#include "cvc_utils.h"
#include "mpi_init.h"
#include "io.h"
#include "propagator_io.h"
#include "contractions_io.h"
#include "read_input_parser.h"
}

#define THREADS_PER_BLOCK 256

/**********************************************
 * reduce a float2 array of length n to one with length 
 *   equal to the number of blocks at launch time 
 *   by blockwise summation
 * - copied from reduce2 in SDK/C/src/reduction/reduction_kernel.cu
 **********************************************/
__global__ void reduce(float2*g_idata, float2*g_odata, unsigned int n) {
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  extern __shared__ float2 sdata[];

  sdata[tid].x = (i < n) ? g_idata[i].x : 0.;
  sdata[tid].y = (i < n) ? g_idata[i].y : 0.;

  __syncthreads();

  for(unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tid < s) {
      sdata[tid].x += sdata[tid + s].x;
      sdata[tid].y += sdata[tid + s].y;
    }
    __syncthreads();
  }
  if (tid == 0) {
    g_odata[blockIdx.x].x = sdata[0].x;
    g_odata[blockIdx.x].y = sdata[0].y;
  }
}
/**********************************************
 * calculate correlation for one lattice site
 * - some kernels will calculate zero
 **********************************************/
__global__ void build_correlator(float2*j_source, float2*j_sink, float2*corr, unsigned int*id_sink, 
                                 unsigned int V, unsigned int mu) {

  unsigned int id_thread = blockIdx.x * blockDim.x + threadIdx.x;

  corr[id_thread].x = j_source[mu*V+id_thread].x * j_sink[mu*V+id_sink[id_thread]].x
                    - j_source[mu*V+id_thread].y * j_sink[mu*V+id_sink[id_thread]].y;
  corr[id_thread].y = j_source[mu*V+id_thread].x * j_sink[mu*V+id_sink[id_thread]].y
                    + j_source[mu*V+id_thread].y * j_sink[mu*V+id_sink[id_thread]].x;
}
/**********************************************
 * initialize a float2 vector of length V to 0.
 **********************************************/
__global__ void init_to_zero(float2*corr, unsigned int V) {

  unsigned int id_thread = blockIdx.x * blockDim.x + threadIdx.x;

  if (id_thread < V) {
    corr[id_thread].x = 0.;
    corr[id_thread].y = 0.;
  }
}

void usage() {
  fprintf(stdout, "Code to calculate correlation of quark-disconnected conserved vector current contractions\n");
  fprintf(stdout, "Usage:    [options]\n");
  fprintf(stdout, "Options: -f <string> input filename [default cvc.input]\n");
  fprintf(stdout, "         -l <uint> spatial size of sublattice [default 2]\n");
  fprintf(stdout, "         -t <uint> temporal size of sublattice [default 2]\n");
  fprintf(stdout, "         -m allow negative entries in the shift vector [default no/0]\n");
#ifdef MPI
  MPI_Abort(MPI_COMM_WORLD, 1);
  MPI_Finalize();
#endif
  exit(0);
}


int main(int argc, char **argv) {
 
  int Tsub = 2; 
  int Lsub = 2; 
  int c, i;
  unsigned int count;
  int filename_set = 0;
  int x0, x1, x2, x3, ip;
  int it, ix, iy, iz, iix;
  int x0b, x0e, x1b, x1e, x2b, x2e, x3b, x3e;
  int sid1, sid2, gid;
  unsigned int *h_ipt_sink=NULL; 
  int include_negative=0, t_start=0, x_start=0, y_start=0, z_start=0;
  size_t nprop=0;
  float *h_data=NULL, *h_swork[48], *h_block_sum=NULL, h_w[2];
  //float *h_swork2=NULL;
  double *h_dwork=NULL;
  float fnorm, r2;
  char filename[100];
  double ratime, retime;
  FILE *ofs=NULL;
  time_t the_time;

  unsigned int threadsPerBlock, blocksPerGrid, blocksPerGridAsThreads;
  unsigned int *d_ipt_sink=NULL; 
  float2 *d_work1=NULL, *d_work2=NULL, *d_work3=NULL, *d_w=NULL;
  float2 *d_block_sum=NULL, *d_block_sum2=NULL, *d_block_sum_ptr1=NULL, *d_block_sum_ptr2=NULL, *d_block_sum_ptr3=NULL;
  unsigned int V4, mu;

  /****************************************
   * initialize the distance vectors
   ****************************************/

  while ((c = getopt(argc, argv, "h?f:l:t:m")) != -1) {
    switch (c) {
    case 'f':
      strcpy(filename, optarg);
      filename_set=1;
      break;
    case 'l':
      Lsub = atoi(optarg);
      fprintf(stdout, "# using Lsub = %d\n", Lsub);
      break;
    case 't':
      Tsub = atoi(optarg);
      fprintf(stdout, "# using Tsub = %d\n", Tsub);
      break;
    case 'm':
      include_negative = 1;
      fprintf(stdout, "# will do negative R_i, too\n");
      break;
    case 'h':
    case '?':
    default:
      usage();
      break;
    }
  }

  /* set the default values */
  if(filename_set==0) strcpy(filename, "cvc.input");
  fprintf(stdout, "# Reading input from file %s\n", filename);
  read_input_parser(filename);

  /* some checks on the input data */
  if((T_global == 0) || (LX==0) || (LY==0) || (LZ==0)) {
    if(g_proc_id==0) fprintf(stderr, "Error, T and L's must be set\n");
    usage();
  }
  if(LX!=LY || LX!=LZ || LY!=LZ) {
    if(g_proc_id==0) fprintf(stderr, "Error, LX, LY and LZ must be mutually equal\n");
    usage();
  }

  if(g_kappa == 0.) {
    if(g_proc_id==0) fprintf(stderr, "Error, kappa should be > 0.n");
    usage();
  }

  fprintf(stdout, "\n**************************************************\n");
  fprintf(stdout, "* jc_ud_p\n* %s", ctime(&the_time));
  fprintf(stdout, "**************************************************\n\n");

  /* initialize fftw */
  T            = T_global;
  L            = LX;
  Tstart       = 0;
  if(!include_negative) {
    FFTW_LOC_VOLUME = Tsub * Lsub*Lsub*Lsub;
  } else {
    FFTW_LOC_VOLUME = (2*Tsub-1) * (2*Lsub-1) * (2*Lsub-1) * (2*Lsub-1);
  }
  fprintf(stdout, "# [%2d] parameters:\n"\
                  "#       T            = %3d\n"\
		  "#       Tstart       = %3d\n"\
		  "#       FFTW_LOC_VOLUME = %8d\n",
		  g_cart_id, T, Tstart, FFTW_LOC_VOLUME);

  if(init_geometry() != 0) {
    fprintf(stderr, "ERROR from init_geometry\n");
    exit(1);
  }

  geometry();

  V4 = (unsigned int)VOLUME;
  /***********************************************
   * set number of threads and blocks
   ***********************************************/
  threadsPerBlock        = THREADS_PER_BLOCK;
  blocksPerGrid          = (V4+threadsPerBlock-1)/threadsPerBlock;
  blocksPerGridAsThreads = ((blocksPerGrid+threadsPerBlock-1)/threadsPerBlock)*threadsPerBlock;
  fprintf(stdout, "# number threads per block: %u\n", threadsPerBlock);
  fprintf(stdout, "# number blocks per grid  : %u\n", blocksPerGrid);
  fprintf(stdout, "# blocksPerGrid as threads: %u\n", blocksPerGridAsThreads);
 
  /****************************************
   * allocate host fields
   ****************************************/
  h_data = (float*)calloc(8*FFTW_LOC_VOLUME, sizeof(float));
  if( h_data==NULL ) { 
    fprintf(stderr, "could not allocate memory for h_data\n");
    exit(3);
  }

  nprop = (size_t)(g_sourceid2 - g_sourceid) / (size_t)g_sourceid_step + 1;
  fprintf(stdout, "# number of stoch. propagators = %lu\n", nprop);

  h_swork[0] = (float*)calloc(nprop*8*(VOLUME+1), sizeof(float));
  if( h_swork[0] == NULL ) { 
    fprintf(stderr, "could not allocate memory for h_swork field\n");
    exit(5);
  }
  for(i=1; i< nprop; i++) {
    h_swork[i] = h_swork[i-1] + 8*(VOLUME+1);
  }

  h_dwork = (double*)calloc(8*VOLUME, sizeof(double));
  if( h_dwork == NULL ) { 
    fprintf(stderr, "could not allocate memory for h_dwork\n");
    exit(15);
  }

  //h_swork2 = (float*)calloc(2*VOLUME, sizeof(float));
  //if( h_swork2 == NULL ) { 
  //  fprintf(stderr, "could not allocate memory for h_swork2\n");
  //  exit(18);
  //}

  h_ipt_sink   = (unsigned int*)calloc(VOLUME, sizeof(unsigned int));
  if(h_ipt_sink==NULL) {
    fprintf(stderr, "could not allocate memory for h_ipt_sink\n");
    exit(16);
  }

  h_block_sum   = (float*)calloc(2*blocksPerGrid, sizeof(float));
  if(h_block_sum==NULL) {
    fprintf(stderr, "could not allocate memory for h_block_sum\n");
    exit(17);
  }

  /***********************************************
   * allocate GPU fields 
   ***********************************************/
  hipMalloc(&d_work1, 4*(VOLUME+1)*sizeof(float2));
  hipMalloc(&d_work2, 4*(VOLUME+1)*sizeof(float2));
  hipMalloc(&d_work3, VOLUME*sizeof(float2));
  hipMalloc(&d_ipt_sink, VOLUME*sizeof(unsigned int));
  hipMalloc(&d_w, sizeof(float2));
  hipMalloc(&d_block_sum, blocksPerGrid*sizeof(float2));
  hipMalloc(&d_block_sum2, blocksPerGrid*sizeof(float2));

  init_to_zero<<<4*blocksPerGrid, threadsPerBlock>>>(d_work1, 4*V4);
  init_to_zero<<<4*blocksPerGrid, threadsPerBlock>>>(d_work1, 4*V4);
  init_to_zero<<<blocksPerGrid, threadsPerBlock>>>(d_work3, V4);
  init_to_zero<<<(blocksPerGrid+threadsPerBlock-1)/threadsPerBlock, threadsPerBlock>>>(d_block_sum, blocksPerGrid);
  init_to_zero<<<(blocksPerGrid+threadsPerBlock-1)/threadsPerBlock, threadsPerBlock>>>(d_block_sum2, blocksPerGrid);
  init_to_zero<<<1, 2>>>(d_w, 2);

  fnorm = 1. / ( (float)nprop * (float)(nprop-1));
  fprintf(stdout, "# fnorm = %16.7e\n", fnorm);

  /***********************************************
   * choose the start values for the entries
   *   of the R-vector
   ***********************************************/
  if(include_negative) {
    t_start = -Tsub+1;
    x_start = -Lsub+1;
    y_start = -Lsub+1;
    z_start = -Lsub+1;
  } else {
    t_start = 0;
    x_start = 0;
    y_start = 0;
    z_start = 0;
  }
  fprintf(stdout, "#\n# t_start=%d, x_start=%d, y_start=%d, z_start=%d\n", 
    t_start, x_start, y_start, z_start);

  /***********************************************
   * start loop on gauge id.s 
   ***********************************************/
  for(gid=g_gaugeid; gid<=g_gaugeid2; gid++) {

    for(ix=0; ix<8*FFTW_LOC_VOLUME; ix++) h_data[ix] = 0.;

    /************************************************
     * read the contracted currents
     ************************************************/
    ratime = clock() / CLOCKS_PER_SEC;
    for(sid1=0; sid1<nprop; sid1++) {
      sprintf(filename, "jc_ud_x.%.4d.%.4d", gid, g_sourceid + sid1*g_sourceid_step);
      if(read_lime_contraction(h_dwork, filename, 4, 0) != 0) {
        fprintf(stderr, "Error, could not read field no. %d\n", sid1);
        exit(15);
      }
      count=0; iix=0;
      for(mu=0; mu<4; mu++) {
        for(ix=0; ix<VOLUME; ix++) {
          h_swork[sid1][iix  ] = (float)(h_dwork[count  ]);
          h_swork[sid1][iix+1] = (float)(h_dwork[count+1]);
          count+=2; iix+=2;
        }
        h_swork[sid1][iix  ] = 0.;
        h_swork[sid1][iix+1] = 0.;
        iix+=2;
      }
      //sprintf(filename, "jc_ud_x.%.4d.%.4d.ascii", gid, g_sourceid+sid1*g_sourceid_step);
      //ofs = fopen(filename, "w");
      //for(i=0; i<4*V4; i++)
      //  fprintf(ofs, "%25.16e%25.16e\n", h_dwork[2*i], h_dwork[2*i+1]);
      //fclose(ofs);
    }
    retime = clock() / CLOCKS_PER_SEC;
    fprintf(stdout, "# time for reading fields: %e seconds\n", retime-ratime);
    /***********************************************
     * start (double) loop on source id pairs
     ***********************************************/
    ratime = (double)clock() / CLOCKS_PER_SEC;
/*    for(sid1=0; sid1<nprop-1; sid1++) { */
    for(sid1=0; sid1<nprop; sid1++) {
      hipMemcpy(d_work1, h_swork[sid1], (VOLUME+1)*8*sizeof(float), hipMemcpyHostToDevice);
/*    for(sid2=sid1+1; sid2<nprop; sid2++) { */
      sid2 = sid1;
      hipMemcpy(d_work2, h_swork[sid2], (VOLUME+1)*8*sizeof(float), hipMemcpyHostToDevice);

      fprintf(stdout, "# processing sid pair (%3d,%3d)\n", sid1, sid2);

      ip = 0;
      for(it=t_start; it<Tsub; it++) {
        x0b = it>=0 ? it : 0;
        x0e = it>=0 ? T  : T+it;
      for(ix=x_start; ix<Lsub; ix++) {
        x1b = ix>=0 ? ix : 0;
        x1e = ix>=0 ? L  : L+ix;
      for(iy=y_start; iy<Lsub; iy++) {
        x2b = iy>=0 ? iy : 0;
        x2e = iy>=0 ? L  : L+iy;
      for(iz=z_start; iz<Lsub; iz++) {
        x3b = iz>=0 ? iz : 0;
        x3e = iz>=0 ? L  : L+iz;
       
        //fprintf(stdout, "# processing shift (%3d,%3d,%3d,%3d)\n", it, ix, iy, iz);
        //fprintf(stdout, "# x0be=(%3d,%3d), x1be=(%3d,%3d), x2be=(%3d,%3d), x3be=(%3d,%3d)\n",
        //  x0b, x0e, x1b, x1e, x2b, x2e, x3b, x3e);
        for(i=0; i<VOLUME; i++) h_ipt_sink[i] = V4;

        for(x0 = x0b; x0 < x0e; x0++) {
        for(x1 = x1b; x1 < x1e; x1++) {
        for(x2 = x2b; x2 < x2e; x2++) {
        for(x3 = x3b; x3 < x3e; x3++) {
          iix = g_ipt[x0][x1][x2][x3];
          h_ipt_sink[iix]   = (unsigned int)g_ipt[x0-it][x1-ix][x2-iy][x3-iz];
        }}}}
        //for(i=0; i<V4; i++) fprintf(stdout, "h_ipt_sink[%d] = %u\n", i, h_ipt_sink[i]);
 
        hipMemcpy(d_ipt_sink, h_ipt_sink, V4*sizeof(unsigned int), hipMemcpyHostToDevice);

        for(mu=0; mu<4; mu++) {
          for(i=0; i<2*blocksPerGrid; i++) h_block_sum[i] = 0.;

          build_correlator<<<blocksPerGrid, threadsPerBlock>>>(d_work1, d_work2, d_work3, d_ipt_sink, V4+1, mu);
          //hipMemcpy(h_swork2, d_work3, 2*V4*sizeof(float), hipMemcpyDeviceToHost);
          //sprintf(filename, "j1xj2.%.2d.%.6d", mu, ip);
          //ofs = fopen(filename, "w");
          //for(i=0; i<V4; i++) {
          //  fprintf(ofs, "%18.9e%18.9e\n", h_swork2[2*i], h_swork2[2*i+1]);
          //}
          //fclose(ofs);
          //h_w[0]=0.; h_w[1]=0.;
          //for(i=0; i<V4; i++) {
          //  h_w[0] += h_swork2[2*i  ];
          //  h_w[1] += h_swork2[2*i+1];
          //}
          //fprintf(stdout, "ord. sum: mu=%3d, ip=%6d, h_w=(%18.9e,%18.9e)\n", mu, ip, h_w[0], h_w[1]);

          reduce<<<blocksPerGrid, threadsPerBlock, threadsPerBlock*sizeof(float2)>>>(d_work3, d_block_sum, V4);
          //hipMemcpy(h_block_sum, d_block_sum, 2*blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);
          //for(i=0; i<blocksPerGrid; i++)
          //  fprintf(stdout, "1st blockSum: %6d%18.9e%18.9e\n", i, h_block_sum[2*i], h_block_sum[2*i+1]);

          /*******************************************************************
           * stepwise reduction of d_block_sum
           *******************************************************************/
          d_block_sum_ptr1 = d_block_sum2;
          d_block_sum_ptr2 = d_block_sum;
          for(count=blocksPerGrid; count>1; count=(count+threadsPerBlock-1)/threadsPerBlock) {
            d_block_sum_ptr3 = d_block_sum_ptr1;
            d_block_sum_ptr1 = d_block_sum_ptr2;
            d_block_sum_ptr2 = d_block_sum_ptr3;
            reduce<<<(count+threadsPerBlock-1)/threadsPerBlock, threadsPerBlock, threadsPerBlock*sizeof(float2)>>>(d_block_sum_ptr1, d_block_sum_ptr2, count);
          }
          h_w[0]=0.; h_w[1]=0.;
          hipMemcpy(h_w, d_block_sum_ptr2, 2*sizeof(float), hipMemcpyDeviceToHost);
          //fprintf(stdout, "blocked sum: mu=%3d, ip=%6d, h_w=%18.9e+%18.9e*1.i\n", mu, ip, h_w[0], h_w[1]);
          h_data[2*(mu*FFTW_LOC_VOLUME+ip)  ] += h_w[0];
          h_data[2*(mu*FFTW_LOC_VOLUME+ip)+1] += h_w[1];
        }
        ip++;
      }}}}
/*    } */ /* of loop on sid2 */
    }  /* of loop on sid1 */

    //for(ix=0; ix<8*FFTW_LOC_VOLUME; ix++) h_data[ix] *= fnorm;
    retime = (double)clock() / CLOCKS_PER_SEC;
    if(g_cart_id == 0) fprintf(stdout, "# time for building correl.: %e seconds\n", retime-ratime);

    /************************************************
     * save results in position space
     ************************************************/
    ratime = (double)clock() / CLOCKS_PER_SEC;
    sprintf(filename, "pi_ud_r.%4d", gid);
    ofs = fopen(filename, "w");
    if (ofs==NULL) {
     fprintf(stderr, "Error, could not open file %s for writing\n", filename);
     exit(9);
    }
    for(mu=0; mu<4; mu++) {
      ip = 0;
      for(it=t_start; it<Tsub; it++) {
        for(ix=x_start; ix<Lsub; ix++) {
        for(iy=y_start; iy<Lsub; iy++) {
        for(iz=z_start; iz<Lsub; iz++) {
          r2 = (double)(ix*ix) + (double)(iy*iy) + (double)(iz*iz);
          fprintf(ofs, "%3d%3d%3d%3d%3d%16.7e%25.16e%25.16e\n", mu, it, ix, iy, iz, r2,
            h_data[_GWI(mu,ip,FFTW_LOC_VOLUME)], h_data[_GWI(mu,ip,FFTW_LOC_VOLUME)+1]);
          ip++;
        }}}
      }
    }
    fclose(ofs);
    retime = (double)clock() / CLOCKS_PER_SEC;
    fprintf(stdout, "# time for writing correl. for file: %e seconds\n", retime-ratime);
  }  /* of loop on gid */

  /***********************************************
   * free the allocated memory, finalize 
   ***********************************************/
  hipFree(d_work1);
  hipFree(d_work2);
  hipFree(d_work3);
  hipFree(d_ipt_sink);
  hipFree(d_w);
  hipFree(d_block_sum);
  hipFree(d_block_sum2);

  free_geometry();
  if(h_dwork     !=NULL) free(h_dwork);
  //if(h_swork2    !=NULL) free(h_swork2);
  if(h_swork[0]  !=NULL) free(h_swork[0]);
  if(h_data      !=NULL) free(h_data);
  if(h_ipt_sink  !=NULL) free(h_ipt_sink);
  if(h_block_sum !=NULL) free(h_block_sum);
  return(0);

}
