#include "hip/hip_runtime.h"
/*********************************************************************************
 * jc_ud_p2.cu
 *
 * Sat Sep 25 16:21:00 CEST 2010
 *
 * PURPOSE:
 * TODO:
 * DONE:
 * CHANGES:
 *********************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#ifdef MPI
#  include <mpi.h>
#  include <fftw_mpi.h>
#else
#  include <fftw.h>
#endif
#include <getopt.h>

#define MAIN_PROGRAM
extern "C" 
{
#include "cvc_complex.h"
#include "cvc_linalg.h"
#include "global.h"
#include "cvc_geometry.h"
#include "cvc_utils.h"
#include "mpi_init.h"
#include "io.h"
#include "propagator_io.h"
#include "contractions_io.h"
#include "read_input_parser.h"
}

#define THREADS_PER_BLOCK 256

/**********************************************
 * reduce a float2 array of length n to one with length 
 *   equal to the number of blocks at launch time 
 *   by blockwise summation
 * - copied from reduce2 in SDK/C/src/reduction/reduction_kernel.cu
 **********************************************/
__global__ void reduce(float2*g_idata, float2*g_odata, unsigned int n) {
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  extern __shared__ float2 sdata[];

  sdata[tid].x = (i < n) ? g_idata[i].x : 0.;
  sdata[tid].y = (i < n) ? g_idata[i].y : 0.;

  __syncthreads();

  for(unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tid < s) {
      sdata[tid].x += sdata[tid + s].x;
      sdata[tid].y += sdata[tid + s].y;
    }
    __syncthreads();
  }
  if (tid == 0) {
    g_odata[blockIdx.x].x = sdata[0].x;
    g_odata[blockIdx.x].y = sdata[0].y;
  }
}
/**********************************************
 * calculate correlation for one lattice site
 * - some kernels will calculate zero
 **********************************************/
__global__ void build_correlator(float2*j_source, float2*j_sink, float2*corr, unsigned int T, unsigned int L, unsigned int Tsub, unsigned int Lsub, unsigned int shift_id, unsigned int mu, int ineg) {

  unsigned int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ int thread_coords[4], shift_coords[4];
  __shared__ unsigned int sink_id;
  __shared__ int x_start[2];
  __shared__ unsigned int L1, L2, L3, Volume;
  __shared__ unsigned int Lsub1, Lsub2, Lsub3;
  __shared__ unsigned int count, ix, iy;
  __shared__ int x0_b, x0_e, x1_b, x1_e, x2_b, x2_e, x3_b, x3_e;
  __shared__ float2 j1x, j2y;

  L1 = L;
  L2 = L1*L1;
  L3 = L2*L1;
  Volume=L3*T;
  x_start[0] = (ineg==0) ? 0 : -Tsub+1;
  x_start[1] = (ineg==0) ? 0 : -Lsub+1;
  
  Lsub1 = Lsub - x_start[1];
  Lsub2 = Lsub1*Lsub1;
  Lsub3 = Lsub2*Lsub1;

  count = thread_id;
  thread_coords[0] = count / L3;
  count -= thread_coords[0]*L3;
  thread_coords[1] = count / L2;
  count -= thread_coords[1]*L2;
  thread_coords[2] = count / L;
  count -= thread_coords[2]*L;
  thread_coords[3] = count;

  count = shift_id;
  shift_coords[0] = count / Lsub3;
  count -= shift_coords[0]*Lsub3;
  shift_coords[1] = count / Lsub2;
  count -= shift_coords[1]*Lsub2;
  shift_coords[2] = count / Lsub1;
  count -= shift_coords[2]*Lsub1;
  shift_coords[3] = count;

  shift_coords[0] += x_start[0];
  shift_coords[1] += x_start[1];
  shift_coords[2] += x_start[1];
  shift_coords[3] += x_start[1];

  x0_b = (shift_coords[0]>=0) ? shift_coords[0] : 0;
  x0_e = T + shift_coords[0] - x0_b;

  x1_b = (shift_coords[1]>=0) ? shift_coords[1] : 0;
  x1_e = L + shift_coords[1] - x1_b;

  x2_b = (shift_coords[2]>=0) ? shift_coords[2] : 0;
  x2_e = L + shift_coords[2] - x2_b;

  x3_b = (shift_coords[3]>=0) ? shift_coords[3] : 0;
  x3_e = L + shift_coords[3] - x3_b;

  sink_id = (thread_coords[0] >= x0_b && thread_coords[0] < x0_e && thread_coords[1] >= x1_b && thread_coords[1] < x1_e && thread_coords[2] >= x2_b && thread_coords[2] < x2_e && thread_coords[3] >= x3_b && thread_coords[3] < x3_e) ? (((thread_coords[0]-shift_coords[0])*L1+ (thread_coords[1]-shift_coords[1]))*L1+ (thread_coords[2]-shift_coords[2]))*L1+ (thread_coords[3]-shift_coords[3]): Volume;

  ix = mu*(Volume+1) + thread_id;
  iy = mu*(Volume+1) + sink_id;
  j1x.x = j_source[ix].x; j1x.y = j_source[ix].y;
  j2y.x = j_sink[iy].x;   j2y.y = j_sink[iy].y;

  corr[thread_id].x = j1x.x * j2y.x - j1x.y * j2y.y;
  corr[thread_id].y = j1x.x * j2y.y + j1x.y * j2y.x;
}
/**********************************************
 * initialize a float2 vector of length V to 0.
 **********************************************/
__global__ void init_to_zero(float2*corr, unsigned int V) {

  unsigned int id_thread = blockIdx.x * blockDim.x + threadIdx.x;

  if (id_thread < V) {
    corr[id_thread].x = 0.;
    corr[id_thread].y = 0.;
  }
}

void usage() {
  fprintf(stdout, "Code to calculate correlation of quark-disconnected conserved vector current contractions\n");
  fprintf(stdout, "Usage:    [options]\n");
  fprintf(stdout, "Options: -f <string> input filename [default cvc.input]\n");
  fprintf(stdout, "         -l <uint> spatial size of sublattice [default 2]\n");
  fprintf(stdout, "         -t <uint> temporal size of sublattice [default 2]\n");
  fprintf(stdout, "         -m allow negative entries in the shift vector [default no/0]\n");
#ifdef MPI
  MPI_Abort(MPI_COMM_WORLD, 1);
  MPI_Finalize();
#endif
  exit(0);
}


int main(int argc, char **argv) {
 
  int Tsub = 2; 
  int Lsub = 2; 
  int c, i;
  unsigned int count, count2;
  int filename_set = 0;
  int ip;
  int it, ix, iy, iz, iix;
  int t_start, x_start, y_start, z_start;
  int sid1, sid2, gid;
  int include_negative=0;
  size_t nprop=0;
  float *h_sdata=NULL, *h_swork[48];
  //float *h_swork2=NULL;
  double *h_dwork=NULL, *h_ddata=NULL;
  float fnorm, r2;
  char filename[100];
  double ratime, retime;
  FILE *ofs=NULL;
  time_t the_time;

  unsigned int threadsPerBlock, blocksPerGrid;
  float2 *d_work1=NULL, *d_work2=NULL, *d_work3=NULL, *d_data=NULL;
  float2 *d_block_sum=NULL, *d_block_sum2=NULL, *d_block_sum_ptr1=NULL, *d_block_sum_ptr2=NULL, *d_block_sum_ptr3=NULL;
  unsigned int V4, mu;

  /****************************************
   * initialize the distance vectors
   ****************************************/

  while ((c = getopt(argc, argv, "h?f:l:t:m")) != -1) {
    switch (c) {
    case 'f':
      strcpy(filename, optarg);
      filename_set=1;
      break;
    case 'l':
      Lsub = atoi(optarg);
      fprintf(stdout, "# using Lsub = %d\n", Lsub);
      break;
    case 't':
      Tsub = atoi(optarg);
      fprintf(stdout, "# using Tsub = %d\n", Tsub);
      break;
    case 'm':
      include_negative = 1;
      fprintf(stdout, "# will do negative R_i, too\n");
      break;
    case 'h':
    case '?':
    default:
      usage();
      break;
    }
  }

  /* get the time stamp */
  the_time = time(NULL);

  /* set the default values */
  if(filename_set==0) strcpy(filename, "cvc.input");
  fprintf(stdout, "# Reading input from file %s\n", filename);
  read_input_parser(filename);

  /* some checks on the input data */
  if((T_global == 0) || (LX==0) || (LY==0) || (LZ==0)) {
    if(g_proc_id==0) fprintf(stderr, "Error, T and L's must be set\n");
    usage();
  }
  if(LX!=LY || LX!=LZ || LY!=LZ) {
    if(g_proc_id==0) fprintf(stderr, "Error, LX, LY and LZ must be mutually equal\n");
    usage();
  }

  if(g_kappa == 0.) {
    if(g_proc_id==0) fprintf(stderr, "Error, kappa should be > 0.n");
    usage();
  }

  fprintf(stdout, "\n**************************************************\n");
  fprintf(stdout, "* jc_ud_p2\n* %s", ctime(&the_time));
  fprintf(stdout, "**************************************************\n\n");

  /* initialize fftw */
  T            = T_global;
  L            = LX;
  Tstart       = 0;
  if(!include_negative) {
    FFTW_LOC_VOLUME = Tsub * Lsub*Lsub*Lsub;
  } else {
    FFTW_LOC_VOLUME = (2*Tsub-1) * (2*Lsub-1) * (2*Lsub-1) * (2*Lsub-1);
  }
  fprintf(stdout, "# [%2d] parameters:\n"\
                  "#       T            = %3d\n"\
		  "#       Tstart       = %3d\n"\
		  "#       FFTW_LOC_VOLUME = %8d\n",
		  g_cart_id, T, Tstart, FFTW_LOC_VOLUME);

  if(init_geometry() != 0) {
    fprintf(stderr, "ERROR from init_geometry\n");
    exit(1);
  }

  geometry();

  V4 = (unsigned int)VOLUME;

  nprop = (size_t)(g_sourceid2 - g_sourceid) / (size_t)g_sourceid_step + 1;
  fprintf(stdout, "# number of stoch. propagators = %lu\n", nprop);

  /***********************************************
   * set number of threads and blocks
   ***********************************************/
  threadsPerBlock        = THREADS_PER_BLOCK;
  blocksPerGrid          = (V4+threadsPerBlock-1)/threadsPerBlock;
  fprintf(stdout, "# number threads per block: %u\n", threadsPerBlock);
  fprintf(stdout, "# number blocks per grid  : %u\n", blocksPerGrid);
 
  /****************************************
   * allocate host fields
   ****************************************/
  h_sdata = (float*)calloc(8*FFTW_LOC_VOLUME, sizeof(float));
  if( h_sdata==NULL ) { 
    fprintf(stderr, "could not allocate memory for h_sdata\n");
    exit(3);
  }

  h_ddata = (double*)calloc(8*FFTW_LOC_VOLUME, sizeof(double));
  if( h_ddata==NULL ) { 
    fprintf(stderr, "could not allocate memory for h_ddata\n");
    exit(3);
  }

  h_swork[0] = (float*)calloc(nprop*8*(VOLUME+1), sizeof(float));
  if( h_swork[0] == NULL ) { 
    fprintf(stderr, "could not allocate memory for h_swork field\n");
    exit(5);
  }
  for(i=1; i< nprop; i++) {
    h_swork[i] = h_swork[i-1] + 8*(VOLUME+1);
  }

  h_dwork = (double*)calloc(8*VOLUME, sizeof(double));
  if( h_dwork == NULL ) { 
    fprintf(stderr, "could not allocate memory for h_dwork\n");
    exit(15);
  }

  //h_swork2 = (float*)calloc(2*VOLUME, sizeof(float));
  //if( h_swork2 == NULL ) { 
  //  fprintf(stderr, "could not allocate memory for h_swork2 field\n");
  //  exit(5);
  //}
  /***********************************************
   * allocate GPU fields 
   ***********************************************/
  hipMalloc(&d_work1, 4*(VOLUME+1)*sizeof(float2));
  hipMalloc(&d_work2, 4*(VOLUME+1)*sizeof(float2));
  hipMalloc(&d_work3, VOLUME*sizeof(float2));
  hipMalloc(&d_data, 4*FFTW_LOC_VOLUME*sizeof(float2));
  hipMalloc(&d_block_sum, blocksPerGrid*sizeof(float2));
  hipMalloc(&d_block_sum2, blocksPerGrid*sizeof(float2));

  init_to_zero<<<4*blocksPerGrid, threadsPerBlock>>>(d_work1, 4*V4);
  init_to_zero<<<4*blocksPerGrid, threadsPerBlock>>>(d_work1, 4*V4);
  init_to_zero<<<blocksPerGrid, threadsPerBlock>>>(d_work3, V4);
  init_to_zero<<<(blocksPerGrid+threadsPerBlock-1)/threadsPerBlock, threadsPerBlock>>>(d_block_sum, blocksPerGrid);
  init_to_zero<<<(blocksPerGrid+threadsPerBlock-1)/threadsPerBlock, threadsPerBlock>>>(d_block_sum2, blocksPerGrid);
  init_to_zero<<<(4*FFTW_LOC_VOLUME+threadsPerBlock-1)/threadsPerBlock, threadsPerBlock>>>(d_data, 4*FFTW_LOC_VOLUME);

  fnorm = 1. / ( (float)nprop * (float)(nprop-1));
  fprintf(stdout, "# fnorm = %16.7e\n", fnorm);

  /***********************************************
   * choose the start values for the entries
   *   of the R-vector
   ***********************************************/
  if(include_negative) {
    t_start = -Tsub+1;
    x_start = -Lsub+1;
    y_start = -Lsub+1;
    z_start = -Lsub+1;
  } else {
    t_start = 0;
    x_start = 0;
    y_start = 0;
    z_start = 0;
  }
  fprintf(stdout, "#\n# t_start=%d, x_start=%d, y_start=%d, z_start=%d\n", 
    t_start, x_start, y_start, z_start);

  /***********************************************
   * start loop on gauge id.s 
   ***********************************************/
  for(gid=g_gaugeid; gid<=g_gaugeid2; gid++) {

    for(ix=0; ix<8*FFTW_LOC_VOLUME; ix++) h_ddata[ix] = 0.;

    /************************************************
     * read the contracted currents
     ************************************************/
    ratime = clock() / CLOCKS_PER_SEC;
    for(sid1=0; sid1<nprop; sid1++) {
      sprintf(filename, "jc_ud_x.%.4d.%.4d", gid, g_sourceid + sid1*g_sourceid_step);
      if(read_lime_contraction(h_dwork, filename, 4, 0) != 0) {
        fprintf(stderr, "Error, could not read field no. %d\n", sid1);
        exit(15);
      }
      count=0; iix=0;
      for(mu=0; mu<4; mu++) {
        for(ix=0; ix<VOLUME; ix++) {
          h_swork[sid1][iix  ] = (float)(h_dwork[count  ]);
          h_swork[sid1][iix+1] = (float)(h_dwork[count+1]);
          count+=2; iix+=2;
        }
        h_swork[sid1][iix  ] = 0.;
        h_swork[sid1][iix+1] = 0.;
        iix+=2;
      }
      //sprintf(filename, "jc_ud_x.%.4d.%.4d.ascii", gid, g_sourceid+sid1*g_sourceid_step);
      //ofs = fopen(filename, "w");
      //for(i=0; i<4*V4; i++)
      //  fprintf(ofs, "%25.16e%25.16e\n", h_dwork[2*i], h_dwork[2*i+1]);
      //fclose(ofs);
    }
    retime = clock() / CLOCKS_PER_SEC;
    fprintf(stdout, "# time for reading fields: %e seconds\n", retime-ratime);
    /***********************************************
     * start (double) loop on source id pairs
     ***********************************************/
    ratime = (double)clock() / CLOCKS_PER_SEC;
    for(sid1=0; sid1<nprop-1; sid1++) {
      hipMemcpy(d_work1, h_swork[sid1], (VOLUME+1)*8*sizeof(float), hipMemcpyHostToDevice);
    for(sid2=sid1+1; sid2<nprop; sid2++) {
      hipMemcpy(d_work2, h_swork[sid2], (VOLUME+1)*8*sizeof(float), hipMemcpyHostToDevice);

      fprintf(stdout, "# processing sid pair (%3d,%3d)\n", sid1, sid2);

      for(mu=0; mu<4; mu++) {
        for(ip=0; ip<FFTW_LOC_VOLUME; ip++) {

          build_correlator<<<blocksPerGrid, threadsPerBlock>>>(d_work1, d_work2, d_work3, T, L, Tsub, Lsub, ip, mu, include_negative);
          //hipMemcpy(h_swork2, d_work3, 2*V4*sizeof(float), hipMemcpyDeviceToHost);
          //sprintf(filename, "j1xj2_2.%.2d.%.6d", mu, ip);
          //ofs = fopen(filename, "w");
          //for(i=0; i<V4; i++) {
          //  fprintf(ofs, "%18.9e%18.9e\n", h_swork2[2*i], h_swork2[2*i+1]);
          //}
          //fclose(ofs);

          reduce<<<blocksPerGrid, threadsPerBlock, threadsPerBlock*sizeof(float2)>>>(d_work3, d_block_sum, V4);

          /*******************************************************************
           * stepwise reduction of d_block_sum
           *******************************************************************/
          d_block_sum_ptr1 = d_block_sum2;
          d_block_sum_ptr2 = d_block_sum;
          for(count=blocksPerGrid;;) {
            d_block_sum_ptr3 = d_block_sum_ptr1;
            d_block_sum_ptr1 = d_block_sum_ptr2;
            d_block_sum_ptr2 = d_block_sum_ptr3;
            count2 = (count+threadsPerBlock-1)/threadsPerBlock;
            if(count2==1) break;
            reduce<<<count2, threadsPerBlock, threadsPerBlock*sizeof(float2)>>>(d_block_sum_ptr1, d_block_sum_ptr2, count);
            count = count2;
          }
          d_block_sum_ptr2 = &(d_data[mu*FFTW_LOC_VOLUME+ip]);
          reduce<<<1, threadsPerBlock, threadsPerBlock*sizeof(float2)>>>(d_block_sum_ptr1, d_block_sum_ptr2, count);
        }
      }
      hipMemcpy(h_sdata, d_data, 8*FFTW_LOC_VOLUME*sizeof(float), hipMemcpyDeviceToHost);
      for(ip=0; ip<8*FFTW_LOC_VOLUME; ip++) h_ddata[ip] += (double)h_sdata[ip];
    }  /* of loop on sid2 */
    }  /* of loop on sid1 */

    //for(ix=0; ix<8*FFTW_LOC_VOLUME; ix++) h_ddata[ix] *= fnorm;
    retime = (double)clock() / CLOCKS_PER_SEC;
    if(g_cart_id == 0) fprintf(stdout, "# time for building correl.: %e seconds\n", retime-ratime);

    /************************************************
     * save results in position space
     ************************************************/
    ratime = (double)clock() / CLOCKS_PER_SEC;
    sprintf(filename, "pi_ud_r2.%4d", gid);
    ofs = fopen(filename, "w");
    if (ofs==NULL) {
     fprintf(stderr, "Error, could not open file %s for writing\n", filename);
     exit(9);
    }
    iix = 0;
    for(mu=0; mu<4; mu++) {
      ip = 0;
      for(it=t_start; it<Tsub; it++) {
        for(ix=x_start; ix<Lsub; ix++) {
        for(iy=y_start; iy<Lsub; iy++) {
        for(iz=z_start; iz<Lsub; iz++) {
          r2 = (double)(ix*ix) + (double)(iy*iy) + (double)(iz*iz);
          fprintf(ofs, "%3d%3d%3d%3d%3d%16.7e%25.16e%25.16e\n", mu, it, ix, iy, iz, r2, h_ddata[iix], h_ddata[iix+1]);
          ip++; iix+=2;
        }}}
      }
    }
    fclose(ofs);
    retime = (double)clock() / CLOCKS_PER_SEC;
    fprintf(stdout, "# time for writing correl. for file: %e seconds\n", retime-ratime);
  }  /* of loop on gid */

  /***********************************************
   * free the allocated memory, finalize 
   ***********************************************/
  hipFree(d_work1);
  hipFree(d_work2);
  hipFree(d_work3);
  hipFree(d_data);
  hipFree(d_block_sum);
  hipFree(d_block_sum2);

  free_geometry();
  if(h_dwork     !=NULL) free(h_dwork);
  if(h_swork[0]  !=NULL) free(h_swork[0]);
  if(h_ddata     !=NULL) free(h_ddata);
  if(h_sdata     !=NULL) free(h_sdata);
  //if(h_swork2    !=NULL) free(h_swork2);
  return(0);

}
