#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <math.h>
#include <getopt.h>

#define MAIN_PROGRAM
extern "C" 
{
#include "lime.h"
#include "cvc_complex.h"
#include "cvc_linalg.h"
#include "global.h"
#include "cvc_geometry.h"
#include "cvc_utils.h"
#include "mpi_init.h"
#include "io.h"
#include "propagator_io.h"
#include "contractions_io.h"
#include "read_input_parser.h"
}

#define THREADS_PER_BLOCK 256
void print_device_properties(struct hipDeviceProp_t p, FILE*ofs);

__device__ void cm_eq_cm_ti_cm_2x2(float2*u, float2*v, float2*w);
__device__ void cm_eq_cm_ti_cm(float2*u, float2*v, float2*w);
__device__ void cm_eq_cm_ti_cm_dag(float2*u, float2*v, float2*w);
__device__ void cm_eq_zero(float2*u);
__device__ void cm_eq_id(float2*u);
__device__ void cm_reconstruct_gaugelink (float2*s_field, float4*g_field_1, float4*g_field_2);
__device__ void re_eq_tr_cm_ti_cm_dag(float *r,  float2*u, float2*v);
__device__ void re_eq_tr_cm(float *r,  float2*u);

__global__ void d_init_geometry(uint4*up, uint4*dn);
__global__ void plaquette(float*plaq, float4*g_field);

__global__ void reconstruct_gauge(float2*rec_gauge, float4*g_field);
__global__ void plaquette(float*plaq, float4*g_field);

__constant__ unsigned int devVolume, devVol3, devT, devL;
float4 *d_gauge_field;
__device__ uint4 *d_iup, *d_idn;
uint4 *d_iup_field, *d_idn_field;

/****************************************************************
 * initialize the next-neighbor fields
 ****************************************************************/
__global__ void d_init_geometry(uint4*up, uint4*dn) {
  unsigned int tid, gid;
  unsigned int x0, x1, x2, x3;
  unsigned int y0, y1, y2, y3;
  uint4 nn;
  unsigned int L, L2, L3, uitmp;
  tid = threadIdx.x;
  gid = blockIdx.x*blockDim.x + threadIdx.x;

  if(gid==0) {
    d_iup = up;
    d_idn = dn;
  }
  __syncthreads();

  L = devL;
  L2 = L*L;
  L3 = L2*L;

  /*************************************/

  x0 = gid / L3;
  uitmp = gid - x0*L3;
  x1 = uitmp / L2;
  uitmp = uitmp - x1*L2;
  x2 = uitmp / L;
  x3 = uitmp - x2*L;

  /*************************************/

  y0 = x0+1;
  y0 = (y0>=devT) ? y0-devT : y0;
  y1=x1; y2=x2; y3=x3;
  nn.x = y0*L3 + y1*L2 + y2*L + y3;

  y1 = x1+1;
  y1 = (y1>=devL) ? y1-devL : y1;
  y0=x0; y2=x2; y3=x3;
  nn.y = y0*L3 + y1*L2 + y2*L + y3;

  y2 = x2+1;
  y2 = (y2>=devL) ? y2-devL : y2;
  y0=x0; y1=x1; y3=x3;
  nn.z = y0*L3 + y1*L2 + y2*L + y3;

  y3 = x3+1;
  y3 = (y3>=devL) ? y3-devL : y3;
  y0=x0; y1=x1; y2=x2;
  nn.w = y0*L3 + y1*L2 + y2*L + y3;

  d_iup[gid] = nn;

  /*************************************/

  y0 = (x0+devT)-1;
  y0 = (y0>=devT) ? y0-devT : y0;
  y1=x1; y2=x2; y3=x3;
  nn.x = y0*L3 + y1*L2 + y2*L + y3;

  y1 = (x1+devL)-1;
  y1 = (y1>=devL) ? y1-devL : y1;
  y0=x0; y2=x2; y3=x3;
  nn.y = y0*L3 + y1*L2 + y2*L + y3;

  y2 = (x2+devL)-1;
  y2 = (y2>=devL) ? y2-devL : y2;
  y0=x0; y1=x1; y3=x3;
  nn.z = y0*L3 + y1*L2 + y2*L + y3;

  y3 = (x3+devL)-1;
  y3 = (y3>=devL) ? y3-devL : y3;
  y0=x0; y1=x1; y2=x2;
  nn.w = y0*L3 + y1*L2 + y2*L + y3;

  d_idn[gid] = nn;

  __syncthreads();
}


/***********************************
 * u = v x w for 2x2 matrices
 ***********************************/
__device__ void cm_eq_cm_ti_cm_2x2(float2*u, float2*v, float2*w) {
  u[0].x = v[0].x*w[0].x - v[0].y*w[0].y + v[1].x*w[2].x - v[1].y*w[2].y;
  u[0].y = v[0].x*w[0].y + v[0].y*w[0].x + v[1].x*w[2].y + v[1].y*w[2].x;
  u[1].x = v[0].x*w[1].x - v[0].y*w[1].y + v[1].x*w[3].x - v[1].y*w[3].y;
  u[1].y = v[0].x*w[1].y + v[0].y*w[1].x + v[1].x*w[3].y + v[1].y*w[3].x;
  u[2].x = v[2].x*w[0].x - v[2].y*w[0].y + v[3].x*w[2].x - v[3].y*w[2].y;
  u[2].y = v[2].x*w[0].y + v[2].y*w[0].x + v[3].x*w[2].y + v[3].y*w[2].x;
  u[3].x = v[2].x*w[1].x - v[2].y*w[1].y + v[3].x*w[3].x - v[3].y*w[3].y;
  u[3].y = v[2].x*w[1].y + v[2].y*w[1].x + v[3].x*w[3].y + v[3].y*w[3].x;
}
/***********************************
 * u = v x w
 ***********************************/
__device__ void cm_eq_cm_ti_cm(float2*u, float2*v, float2*w) {
  u[0].x = v[0].x*w[0].x - v[0].y*w[0].y + v[1].x*w[3].x - v[1].y*w[3].y + v[2].x*w[6].x - v[2].y*w[6].y;
  u[0].y = v[0].x*w[0].y + v[0].y*w[0].x + v[1].x*w[3].y + v[1].y*w[3].x + v[2].x*w[6].y + v[2].y*w[6].x;
  u[1].x = v[0].x*w[1].x - v[0].y*w[1].y + v[1].x*w[4].x - v[1].y*w[4].y + v[2].x*w[7].x - v[2].y*w[7].y;
  u[1].y = v[0].x*w[1].y + v[0].y*w[1].x + v[1].x*w[4].y + v[1].y*w[4].x + v[2].x*w[7].y + v[2].y*w[7].x;
  u[2].x = v[0].x*w[2].x - v[0].y*w[2].y + v[1].x*w[5].x - v[1].y*w[5].y + v[2].x*w[8].x - v[2].y*w[8].y;
  u[2].y = v[0].x*w[2].y + v[0].y*w[2].x + v[1].x*w[5].y + v[1].y*w[5].x + v[2].x*w[8].y + v[2].y*w[8].x;

  u[3].x = v[3].x*w[0].x - v[3].y*w[0].y + v[4].x*w[3].x - v[4].y*w[3].y + v[5].x*w[6].x - v[5].y*w[6].y;
  u[3].y = v[3].x*w[0].y + v[3].y*w[0].x + v[4].x*w[3].y + v[4].y*w[3].x + v[5].x*w[6].y + v[5].y*w[6].x;
  u[4].x = v[3].x*w[1].x - v[3].y*w[1].y + v[4].x*w[4].x - v[4].y*w[4].y + v[5].x*w[7].x - v[5].y*w[7].y;
  u[4].y = v[3].x*w[1].y + v[3].y*w[1].x + v[4].x*w[4].y + v[4].y*w[4].x + v[5].x*w[7].y + v[5].y*w[7].x;
  u[5].x = v[3].x*w[2].x - v[3].y*w[2].y + v[4].x*w[5].x - v[4].y*w[5].y + v[5].x*w[8].x - v[5].y*w[8].y;
  u[5].y = v[3].x*w[2].y + v[3].y*w[2].x + v[4].x*w[5].y + v[4].y*w[5].x + v[5].x*w[8].y + v[5].y*w[8].x;

  u[6].x = v[6].x*w[0].x - v[6].y*w[0].y + v[7].x*w[3].x - v[7].y*w[3].y + v[8].x*w[6].x - v[8].y*w[6].y;
  u[6].y = v[6].x*w[0].y + v[6].y*w[0].x + v[7].x*w[3].y + v[7].y*w[3].x + v[8].x*w[6].y + v[8].y*w[6].x;
  u[7].x = v[6].x*w[1].x - v[6].y*w[1].y + v[7].x*w[4].x - v[7].y*w[4].y + v[8].x*w[7].x - v[8].y*w[7].y;
  u[7].y = v[6].x*w[1].y + v[6].y*w[1].x + v[7].x*w[4].y + v[7].y*w[4].x + v[8].x*w[7].y + v[8].y*w[7].x;
  u[8].x = v[6].x*w[2].x - v[6].y*w[2].y + v[7].x*w[5].x - v[7].y*w[5].y + v[8].x*w[8].x - v[8].y*w[8].y;
  u[8].y = v[6].x*w[2].y + v[6].y*w[2].x + v[7].x*w[5].y + v[7].y*w[5].x + v[8].x*w[8].y + v[8].y*w[8].x;
}

/***********************************
 * u = v x w^dagger
 ***********************************/
__device__ void cm_eq_cm_ti_cm_dag(float2*u, float2*v, float2*w) {
  u[0].x =  v[0].x*w[0].x + v[0].y*w[0].y + v[1].x*w[3].x + v[1].y*w[3].y + v[2].x*w[6].x + v[2].y*w[6].y;
  u[0].y = -v[0].x*w[0].y + v[0].y*w[0].x - v[1].x*w[3].y + v[1].y*w[3].x - v[2].x*w[6].y + v[2].y*w[6].x;
  u[1].x =  v[0].x*w[1].x + v[0].y*w[1].y + v[1].x*w[4].x + v[1].y*w[4].y + v[2].x*w[7].x + v[2].y*w[7].y;
  u[1].y = -v[0].x*w[1].y + v[0].y*w[1].x - v[1].x*w[4].y + v[1].y*w[4].x - v[2].x*w[7].y + v[2].y*w[7].x;
  u[2].x =  v[0].x*w[2].x + v[0].y*w[2].y + v[1].x*w[5].x + v[1].y*w[5].y + v[2].x*w[8].x + v[2].y*w[8].y;
  u[2].y = -v[0].x*w[2].y + v[0].y*w[2].x - v[1].x*w[5].y + v[1].y*w[5].x - v[2].x*w[8].y + v[2].y*w[8].x;

  u[3].x =  v[3].x*w[0].x + v[3].y*w[0].y + v[4].x*w[3].x + v[4].y*w[3].y + v[5].x*w[6].x + v[5].y*w[6].y;
  u[3].y = -v[3].x*w[0].y + v[3].y*w[0].x - v[4].x*w[3].y + v[4].y*w[3].x - v[5].x*w[6].y + v[5].y*w[6].x;
  u[4].x =  v[3].x*w[1].x + v[3].y*w[1].y + v[4].x*w[4].x + v[4].y*w[4].y + v[5].x*w[7].x + v[5].y*w[7].y;
  u[4].y = -v[3].x*w[1].y + v[3].y*w[1].x - v[4].x*w[4].y + v[4].y*w[4].x - v[5].x*w[7].y + v[5].y*w[7].x;
  u[5].x =  v[3].x*w[2].x + v[3].y*w[2].y + v[4].x*w[5].x + v[4].y*w[5].y + v[5].x*w[8].x + v[5].y*w[8].y;
  u[5].y = -v[3].x*w[2].y + v[3].y*w[2].x - v[4].x*w[5].y + v[4].y*w[5].x - v[5].x*w[8].y + v[5].y*w[8].x;

  u[6].x =  v[6].x*w[0].x + v[6].y*w[0].y + v[7].x*w[3].x + v[7].y*w[3].y + v[8].x*w[6].x + v[8].y*w[6].y;
  u[6].y = -v[6].x*w[0].y + v[6].y*w[0].x - v[7].x*w[3].y + v[7].y*w[3].x - v[8].x*w[6].y + v[8].y*w[6].x;
  u[7].x =  v[6].x*w[1].x + v[6].y*w[1].y + v[7].x*w[4].x + v[7].y*w[4].y + v[8].x*w[7].x + v[8].y*w[7].y;
  u[7].y = -v[6].x*w[1].y + v[6].y*w[1].x - v[7].x*w[4].y + v[7].y*w[4].x - v[8].x*w[7].y + v[8].y*w[7].x;
  u[8].x =  v[6].x*w[2].x + v[6].y*w[2].y + v[7].x*w[5].x + v[7].y*w[5].y + v[8].x*w[8].x + v[8].y*w[8].y;
  u[8].y = -v[6].x*w[2].y + v[6].y*w[2].x - v[7].x*w[5].y + v[7].y*w[5].x - v[8].x*w[8].y + v[8].y*w[8].x;
}

/***********************************
 * set u to zero matrix
 ***********************************/
__device__ void cm_eq_zero(float2*u) {
  u[0].x = 0.; u[0].y = 0.;
  u[1].x = 0.; u[1].y = 0.;
  u[2].x = 0.; u[2].y = 0.;
  u[3].x = 0.; u[3].y = 0.;
  u[4].x = 0.; u[4].y = 0.;
  u[5].x = 0.; u[5].y = 0.;
  u[6].x = 0.; u[6].y = 0.;
  u[7].x = 0.; u[7].y = 0.;
  u[8].x = 0.; u[8].y = 0.;
}

/***********************************
 * set u to identity matrix
 ***********************************/
__device__ void cm_eq_id(float2*u) {
  u[0].x = 1.; u[0].y = 0.;
  u[1].x = 0.; u[1].y = 0.;
  u[2].x = 0.; u[2].y = 0.;
  u[3].x = 0.; u[3].y = 0.;
  u[4].x = 1.; u[4].y = 0.;
  u[5].x = 0.; u[5].y = 0.;
  u[6].x = 0.; u[6].y = 0.;
  u[7].x = 0.; u[7].y = 0.;
  u[8].x = 1.; u[8].y = 0.;
}

/*********************************************
 * kernel to reconstruct the gauge field
 *   from the compressed version
 *********************************************/
__device__ void cm_reconstruct_gaugelink (float2*s_field, float4*g_field_1, float4*g_field_2) {

  __shared__ float ftmp, ftmp2;
  __shared__ float v0x, v0y, v1x, v1y, v2x, v2y, v3x, v3y;
  __shared__ float a1x, a1y, c1x, c1y;
  __shared__ float g1x, g1y, g1z, g1w, g2x, g2y, g2z, g2w;

  g1x = g_field_1[0].x;
  g1y = g_field_1[0].y;
  g1z = g_field_1[0].z;
  g1w = g_field_1[0].w;
  g2x = g_field_2[0].x;
  g2y = g_field_2[0].y;
  g2z = g_field_2[0].z;
  g2w = g_field_2[0].w;

  ftmp = g1x*g1x + g1y*g1y +g1z*g1z +g1w*g1w; // this is N^2
  ftmp2 = sqrtf(1. - ftmp);

  a1x = ftmp2*cosf(g2z);                  
  a1y = ftmp2*sinf(g2z);
  ftmp = 1./ftmp;

  v0x = -g1z; 
  v0y =  g1w;  
  v1x =  g1x; 
  v1y = -g1y;
  v2x = -(a1x*g1x + a1y*g1y); 
  v2y = -(a1x*g1y - a1y*g1x);
  v3x = -(a1x*g1z + a1y*g1w); 
  v3y = -(a1x*g1w - a1y*g1z);

  ftmp2 = sqrtf( 1. - ( a1x*a1x + a1y*a1y + g2x*g2x + g2y*g2y ) );
  c1x = cosf(g2w)*ftmp2; 
  c1y = sinf(g2w)*ftmp2;

  s_field[0].x = a1x;
  s_field[0].y = a1y;
  s_field[1].x = g1x;
  s_field[1].y = g1y;
  s_field[2].x = g1z;
  s_field[2].y = g1w;
  s_field[3].x = g2x;
  s_field[3].y = g2y;
  s_field[4].x =  c1x*v0x + c1y*v0y + g2x*v2x - g2y*v2y;
  s_field[4].y =  c1x*v0y - c1y*v0x + g2x*v2y + g2y*v2x;
  s_field[4].x *= ftmp;
  s_field[4].y *= ftmp;
  s_field[5].x =  c1x*v1x + c1y*v1y + g2x*v3x - g2y*v3y;
  s_field[5].y =  c1x*v1y - c1y*v1x + g2x*v3y + g2y*v3x;
  s_field[5].x *= ftmp;
  s_field[5].y *= ftmp;
  s_field[6].x = c1x;
  s_field[6].y = c1y;
  s_field[7].x = -g2x*v0x - g2y*v0y + c1x*v2x - c1y*v2y;
  s_field[7].y = -g2x*v0y + g2y*v0x + c1x*v2y + c1y*v2x;
  s_field[7].x *= ftmp;
  s_field[7].y *= ftmp;
  s_field[8].x = -g2x*v1x - g2y*v1y + c1x*v3x - c1y*v3y;
  s_field[8].y = -g2x*v1y + g2y*v1x + c1x*v3y + c1y*v3x;
  s_field[8].x *= ftmp;
  s_field[8].y *= ftmp;
} 

/********************************************************************
 * calculate Re ( Tr [ u x v^dagger ] )
 ********************************************************************/
__device__ void re_eq_tr_cm_ti_cm_dag(float r[1],  float2 u[9], float2 v[9]) {
  float tmp;
  tmp  = u[0].x * v[0].x;
  tmp += u[0].y * v[0].y;
  tmp += u[1].x * v[1].x;
  tmp += u[1].y * v[1].y;
  tmp += u[2].x * v[2].x;
  tmp += u[2].y * v[2].y;
  /*
  tmp += u[3].x * v[3].x;
  tmp += u[3].y * v[3].y;
  tmp += u[4].x * v[4].x;
  tmp += u[4].y * v[4].y;
  tmp += u[5].x * v[5].x;
  tmp += u[5].y * v[5].y;
  tmp += u[6].x * v[6].x;
  tmp += u[6].y * v[6].y;
  tmp += u[7].x * v[7].x;
  tmp += u[7].y * v[7].y;
  tmp += u[8].x * v[8].x;
  tmp += u[8].y * v[8].y;
  */
  r[0] = tmp;
}

/********************************************************************
 * calculate Re ( Tr [ u x v^dagger ] )
 ********************************************************************/
__device__ void re_eq_tr_cm(float *r,  float2*u) {
  __shared__ float tmp;
  tmp  = u[0].x;
  tmp += u[4].x;
  tmp += u[8].x;
  r[0] = tmp;
}

/****************************************************************
 * reconstruct the gauge field to a global device memory
 ****************************************************************/
__global__ void reconstruct_gauge(float2*rec_gauge, float4*g_field) {
  unsigned int tid, gid;
  unsigned int uitmp;

  tid = threadIdx.x;
  gid = blockIdx.x*blockDim.x + threadIdx.x;
  uitmp = (devT-1)*devVol3;

  /* reconstruct the spatial links at x */
  cm_eq_id(rec_gauge+36*gid);
  cm_reconstruct_gaugelink(rec_gauge+36*gid+ 9, g_field+gid,             g_field+gid+  devVolume);
  cm_reconstruct_gaugelink(rec_gauge+36*gid+18, g_field+gid+2*devVolume, g_field+gid+3*devVolume);
  cm_reconstruct_gaugelink(rec_gauge+36*gid+27, g_field+gid+4*devVolume, g_field+gid+5*devVolume);
  if(gid >= uitmp) {
    cm_reconstruct_gaugelink(rec_gauge+36*gid,   (g_field+(6*devVolume + (gid-uitmp))), (g_field+(6*devVolume + (gid-uitmp)))+devVol3);
  }
}

/****************************************************************
 * calculate the plaquette
 ****************************************************************/
__global__ void plaquette(float*plaq, float4*g_field) {
  unsigned int tid, gid;
  unsigned int uitmp;
  unsigned int xp0, xp1, xp2, xp3;
  float ftmp[1];
  float2 g0[9], g1[9], g2[9], g3[9], g4[9], u[9], v[9], w[9];

  extern __shared__ float plaq_field[];

  tid = threadIdx.x;
  gid = blockIdx.x*blockDim.x + threadIdx.x;
  xp0 = d_iup[gid].x;
  xp1 = d_iup[gid].y;
  xp2 = d_iup[gid].z;
  xp3 = d_iup[gid].w;

  plaq_field[tid] = 0.;
  /* reconstruct the spatial links at x */
  cm_reconstruct_gaugelink(g0, g_field+gid,             g_field+gid+  devVolume);
  cm_reconstruct_gaugelink(g1, g_field+gid+2*devVolume, g_field+gid+3*devVolume);
  cm_reconstruct_gaugelink(g2, g_field+gid+4*devVolume, g_field+gid+5*devVolume);

  /* U_2 (x+1) */
  cm_reconstruct_gaugelink(g3, g_field+xp1+2*devVolume, g_field+xp1+3*devVolume);
  /* U_1 (x+3) */
  cm_reconstruct_gaugelink(g4, g_field+xp2,             g_field+xp3+  devVolume);
  cm_eq_cm_ti_cm(u, g0, g3);
  cm_eq_cm_ti_cm(v, g1, g4);
  cm_eq_cm_ti_cm(w, u, v);
  //re_eq_tr_cm(ftmp, w);
  plaq_field[tid] += ftmp[0];

//  re_eq_tr_cm(&ftmp, g3);
//  plaq_field[tid] += ftmp;

//  re_eq_tr_cm(&ftmp, g4);
//  plaq_field[tid] += ftmp;
//  re_eq_tr_cm(&ftmp, u);
//  plaq_field[tid] += ftmp;

/*
  cm_reconstruct_gaugelink(g3, g_field+xpn.y+4*devVolume, devVolume);
  cm_reconstruct_gaugelink(g4, g_field+xpn.w, devVolume);
  cm_eq_cm_ti_cm(u, g0, g3);
  cm_eq_cm_ti_cm(v, g2, g4);
  re_eq_tr_cm_ti_cm_dag(&ftmp, u, v);
  plaq_field[tid] += ftmp;

  cm_reconstruct_gaugelink(g3, g_field+xpn.z+4*devVolume, devVolume);
  cm_reconstruct_gaugelink(g4, g_field+xpn.w+2*devVolume, devVolume);
  cm_eq_cm_ti_cm(u, g1, g3);
  cm_eq_cm_ti_cm(v, g2, g4);
  re_eq_tr_cm_ti_cm_dag(&ftmp, u, v);
  plaq_field[tid] += ftmp;
*/
/*
  uitmp = (devT-1)*devVol3;
  if(gid>=uitmp) {

    cm_reconstruct_gaugelink(w, g_field+6*devVolume + gid - uitmp, devVol3);

    cm_reconstruct_gaugelink(g4, g_field+6*devVolume + xpn.y - uitmp, devVol3);
    cm_reconstruct_gaugelink(g3, g_field+xpn.x, devVolume);
    cm_eq_cm_ti_cm(u, w, g3);
    cm_eq_cm_ti_cm(v, g0, g4);
    re_eq_tr_cm_ti_cm_dag(&ftmp, u, v);
    plaq_field[tid] += ftmp;

    cm_reconstruct_gaugelink(g4, g_field+6*devVolume + xpn.z - uitmp, devVol3);
    cm_reconstruct_gaugelink(g3, g_field+xpn.x+2*devVolume, devVolume);
    cm_eq_cm_ti_cm(u, w, g3);
    cm_eq_cm_ti_cm(v, g1, g4);
    re_eq_tr_cm_ti_cm_dag(&ftmp, u, v);
    plaq_field[tid] += ftmp;

    cm_reconstruct_gaugelink(g4, g_field+6*devVolume + xpn.w - uitmp, devVol3);
    cm_reconstruct_gaugelink(g3, g_field+xpn.x+4*devVolume, devVolume);
    cm_eq_cm_ti_cm(u, w, g3);
    cm_eq_cm_ti_cm(v, g2, g4);
    re_eq_tr_cm_ti_cm_dag(&ftmp, u, v);
    plaq_field[tid] += ftmp;
  } else {
    cm_reconstruct_gaugelink(g3, g_field+xpn.x, devVolume);
    re_eq_tr_cm_ti_cm_dag(&ftmp, g3, g0);
    plaq_field[tid] += ftmp;

    cm_reconstruct_gaugelink(g3, g_field+xpn.x+2*devVolume, devVolume);
    re_eq_tr_cm_ti_cm_dag(&ftmp, g3, g1);
    plaq_field[tid] += ftmp;

    cm_reconstruct_gaugelink(g3, g_field+xpn.x+4*devVolume, devVolume);
    re_eq_tr_cm_ti_cm_dag(&ftmp, g3, g2);
    plaq_field[tid] += ftmp;
  }
*/

  __syncthreads();
  for(unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tid < s) { plaq_field[tid] += plaq_field[tid + s]; }
    __syncthreads();
  }
  //if (tid == 0) { plaq[blockIdx.x] = plaq_field[0]; } 
  if (tid == 0) { plaq[blockIdx.x] = (float)xp1; } 
}


/**********************************************************************************
 **********************************************************************************
 **
 ** end of device function declaration / begin of main function
 **
 **********************************************************************************
 **********************************************************************************/

void usage(void) {
  fprintf(stdout, "# Programme; exit\n");
  exit(0);
}

int main (int argc, char *argv[]) {

  int status, c;
  int num_fields = 0;
  int filename_set = 0;
  int it, ix, iix, count, itmp, itmp2, i, j;
  int VOL3;
  unsigned int uitmp, *nn_field;
  unsigned int threadsPerBlock, blocksPerGrid;
  float **spinor_field_flt=NULL, *gauge_field_flt=NULL;
  double *gauge_transform=NULL, *gauge_aux=NULL, U_[18];
  float *h_plaq_field;
  float *gauge_aux2=NULL;
  double plaq, dtmp;
  double ratime, retime;
  char filename[400];
  void *vptr;

  hipError_t cuderr;
  int dev_num;
  struct hipDeviceProp_t *dev_prop;
  float2 *d_gauge_rec;
  float *d_plaq_field;

  /****************************************
   * initialize the distance vectors
   ****************************************/

  while ((c = getopt(argc, argv, "h?f:")) != -1) {
    switch (c) {
      case 'f':
        strcpy(filename, optarg);
        filename_set=1;
        break;
      case 'h':
      case '?':
      default:
        usage();
        break;
    }
  }

  g_the_time = time(NULL);


  mpi_init(argc, argv);

  /* set the default values */
  if(filename_set==0) strcpy(filename, "cvc.input");
  fprintf(stdout, "# Reading input from file %s\n", filename);
  read_input_parser(filename);

  /* some checks on the input data */
  if((T_global == 0) || (LX==0) || (LY==0) || (LZ==0)) {
    if(g_proc_id==0) fprintf(stderr, "Error, T and L's must be set\n");
    usage();
  }

  if(LX!=LY || LX!=LZ || LY!=LZ) {
    if(g_proc_id==0) fprintf(stderr, "Error, LX, LY and LZ must be mutually equal\n");
    usage();
  }

  VOL3 = LX*LY*LZ;
  T = T_global;

  status = init_geometry();
  if(status != 0) {
    fprintf(stderr, "Error from init_geometry, status was %d\n", status);
    exit(1);
  }

  geometry();

  /***************************************
   * try device management
   ***************************************/
  cuderr = hipGetDeviceCount(&dev_num);
  fprintf(stdout, "\n# found %d devices\n", dev_num);
  dev_prop = (struct hipDeviceProp_t*)malloc(dev_num*sizeof(struct hipDeviceProp_t));
  if(dev_prop==NULL) {
    fprintf(stderr, "Error, could not alloc dev_prop\n");
    exit(109);
  }
  for(i=0; i<dev_num; i++) {
    cuderr = hipGetDeviceProperties(dev_prop+i, i);
    print_device_properties(dev_prop[i], stdout);
  }
  free(dev_prop);

  cuderr = hipSetDevice (0);
  if (cuderr == hipErrorSetOnActiveProcess) {
    hipGetDevice(&itmp);
    fprintf(stderr, "Error, could not set device 0, already using device %d\n", itmp);
  }

  /***********************************************
   * set number of threads and blocks
   ***********************************************/
  threadsPerBlock = THREADS_PER_BLOCK;
  blocksPerGrid   = (VOLUME+threadsPerBlock-1)/threadsPerBlock;
  fprintf(stdout, "# number threads per block: %u\n", threadsPerBlock);
  fprintf(stdout, "# number blocks per grid  : %u\n", blocksPerGrid);

  /************************************
   * initialise device constants
   ************************************/
  uitmp = (unsigned int)T;
  if( (cuderr = hipMemcpyToSymbol(HIP_SYMBOL("devT"), &uitmp, sizeof(unsigned int))) != hipSuccess) {
    fprintf(stderr, "Error, could not set devT\n");
    exit(113);
  }

  uitmp = (unsigned int)LX;
  if( (cuderr = hipMemcpyToSymbol(HIP_SYMBOL("devL"), &uitmp, sizeof(unsigned int))) != hipSuccess) {
    fprintf(stderr, "Error, could not set devL\n");
    exit(113);
  }
  uitmp =(unsigned int)VOLUME;
  if( (cuderr = hipMemcpyToSymbol(HIP_SYMBOL("devVolume"), &uitmp, sizeof(unsigned int))) != hipSuccess) {
    fprintf(stderr, "Error, could not set devVolume\n");
    exit(113);
  }
  uitmp =(unsigned int)VOL3;
  if( (cuderr = hipMemcpyToSymbol(HIP_SYMBOL("devVol3"), &uitmp, sizeof(unsigned int))) != hipSuccess) {
    fprintf(stderr, "Error, could not set devVol3\n");
    exit(113);
  }

  /************************************************
   * allocate memory for the nn fields on device
   ************************************************/
  uitmp = VOLUME*sizeof(uint4);
  cuderr = hipMalloc(&d_iup_field, uitmp);
  if(cuderr != hipSuccess) {
    fprintf(stderr, "Error, could not allocate mem on device\n");
    exit(110);
  }

  cuderr = hipMalloc(&d_idn_field, uitmp);
  if(cuderr != hipSuccess) {
    fprintf(stderr, "Error, could not allocate mem on device\n");
    exit(111);
  }

  d_init_geometry<<<blocksPerGrid, threadsPerBlock>>>(d_iup_field, d_idn_field);

  /*********************************************************************************
   **                        end of initialization part                           **
   *********************************************************************************/

  /* read the gauge field */
  alloc_gauge_field_dbl(&g_gauge_field, 72*VOLUMEPLUSRAND);
  sprintf(filename, "%s.%.4d", gaugefilename_prefix, Nconf);
  if(g_cart_id==0) fprintf(stdout, "# reading gauge field from file %s\n", filename);
  read_lime_gauge_field_doubleprec(filename);
#ifdef MPI
  xchange_gauge();
#endif
  /* measure the plaquette */
  plaquette(&plaq);
  if(g_cart_id==0) fprintf(stdout, "# measured plaquette value: %25.16e\n", plaq);

  /* alloc gauge transform field */
  alloc_gauge_field_dbl(&gauge_transform, VOLUME*18);
  if(gauge_transform == NULL) {
    fprintf(stderr, "Error, could not alloc gauge transform field\n");
    exit(16);
  }

  set_temporal_gauge(gauge_transform);
  alloc_gauge_field_dbl(&gauge_aux, VOLUME*72);
  apply_gauge_transform(gauge_aux, gauge_transform, g_gauge_field);

  plaquette2(&plaq, gauge_aux);
  if(g_cart_id==0) fprintf(stdout, "# measured plaquette value after gauge transform: %25.16e\n", plaq);

  alloc_gauge_field_flt(&gauge_field_flt, 8*(3*T+1)*VOL3);
  compress_gauge(gauge_field_flt, gauge_aux);

  /************************************************
   * allocate memory for the gauge field on device
   ************************************************/
 
  uitmp = ( 6*(unsigned int)VOLUME+2*(unsigned int)VOL3 )*sizeof(float4);
  cuderr = hipMalloc(&d_gauge_field, uitmp);
  if(cuderr != hipSuccess) {
    fprintf(stderr, "Error, could not allocate mem on device\n");
    exit(112);
  }
  if( (cuderr = hipMemcpy(d_gauge_field, gauge_field_flt, uitmp, hipMemcpyHostToDevice)) != hipSuccess ) {
    fprintf(stderr, "Error, could not memcpy gauge field to device\n");
    exit(115);
  }

  uitmp = blocksPerGrid * sizeof(float);
  if( (cuderr = hipMalloc(&d_plaq_field, uitmp)) != hipSuccess ) {
    fprintf(stderr, "Error, could not alloc field on device\n");
    exit(125);
  }
  if( (h_plaq_field = (float*)malloc(uitmp))==NULL ) {
    fprintf(stderr, "Error, could not alloc field on host\n");
    exit(16);
  }

  plaquette<<<blocksPerGrid, threadsPerBlock, uitmp>>>(d_plaq_field, d_gauge_field);
  if( (cuderr=hipMemcpy(h_plaq_field, d_plaq_field, uitmp, hipMemcpyDeviceToHost))!=hipSuccess){
    fprintf(stderr, "Error, could not memcpy field from device to host\n");
    exit(127);
  }
  for(i=0; i<blocksPerGrid; i++) fprintf(stdout, "# plaq(%d) = %25.16e\n", i, h_plaq_field[i]);
  for(i=1; i<blocksPerGrid; i++) h_plaq_field[0] += h_plaq_field[i];
  fprintf(stdout, "# plaq as measured on device: %25.16e\n", h_plaq_field[0]);
  free(h_plaq_field);
  hipFree(d_plaq_field);


  /********************************************************************************
   ********************************************************************************
   **
   ** free and finalize
   **
   ********************************************************************************
   ********************************************************************************/
  hipFree(d_iup_field);
  hipFree(d_idn_field);


  fprintf(stderr, "\n# %s# end of run\n", ctime(&g_the_time));
  fflush(stderr);

  fprintf(stdout, "\n# %s# end of run\n", ctime(&g_the_time));
  fflush(stdout);

  return(0);

}


void print_device_properties (struct hipDeviceProp_t p, FILE*ofs) {

  fprintf(ofs, "\n# device properties:\n");
  fprintf(ofs, "# device name: %s\n", p.name);
  fprintf(ofs, "# device global memory: %u\n", p.totalGlobalMem);
  fprintf(ofs, "# device no. of shared memory per block: %u\n", p.sharedMemPerBlock);
  fprintf(ofs, "# device no. of registers per block: %d\n", p.regsPerBlock);
  fprintf(ofs, "# device warp size: %d\n", p.warpSize);
  fprintf(ofs, "# device memory pitch: %u\n", p.memPitch);
  fprintf(ofs, "# device max. no. of threads per block: %d\n", p.maxThreadsPerBlock);
  fprintf(ofs, "# device max. no. of thread dimensions: (%d, %d, %d)\n", 
      p.maxThreadsDim[0], p.maxThreadsDim[1], p.maxThreadsDim[2]);
  fprintf(ofs, "# device maximal grid size: (%d, %d, %d)\n\n", 
      p.maxGridSize[0], p.maxGridSize[1], p.maxGridSize[2]);
  fflush(ofs);

}
